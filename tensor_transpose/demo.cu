
#include <hip/hip_runtime.h>
#include <cassert>
#include <chrono>
#include <fstream>
#include <iostream>
#include <random>


// ============================================================================
// SETTINGS
const int  NUM_REPS     = 100;
const bool ENABLE_PRINT = false;

// const int T       = 8;
// const int TILE_X  = T;
// const int TILE_Y  = T;
// const int TILE_Z  = 1;
// const int BLOCK_X = T;  // dim2
// const int BLOCK_Y = T;  // dim1
// const int BLOCK_Z = T;  // dim0

// const int DIMENSION[] = {64, 64, 64};
// // const int PERMUTATION[] = {1, 2, 0};
// const int PERMUTATION[] = {0, 2, 1};


const int T       = 16;
const int TILE_X  = T;
const int TILE_Y  = T;
const int TILE_Z  = 1;
const int BLOCK_X = T;  // dim2
const int BLOCK_Y = T;  // dim1
const int BLOCK_Z = 1;  // dim0

const int DIMENSION[]   = {64, 64, 64};
const int PERMUTATION[] = {0, 2, 1};


// ============================================================================
// CUDA SECTION
inline hipError_t CHECK_CUDA(hipError_t result) {
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
}

__global__ void copy_kernel(const float* d_input, float* d_output, int dim0,
                            int dim1, int dim2, int p0, int p1, int p2) {
    int idx2 = blockIdx.x * blockDim.x + threadIdx.x;
    int idx1 = blockIdx.y * blockDim.y + threadIdx.y;
    int idx0 = blockIdx.z * blockDim.z + threadIdx.z;

    if (idx0 < dim0 && idx1 < dim1 && idx2 < dim2) {
        int iIndex       = (idx0 * dim1 * dim2) + (idx1 * dim2) + idx2;
        d_output[iIndex] = d_input[iIndex];
    }
}
__global__ void copy_shm_kernel(const float* d_input, float* d_output, int dim0,
                                int dim1, int dim2, int p0, int p1, int p2) {
    __shared__ float buffer[TILE_Z][TILE_Y][TILE_X + 1];

    int iDim[3] = {dim0, dim1, dim2};
    int i       = blockIdx.z * TILE_Z + threadIdx.z;
    int j       = blockIdx.y * TILE_Y + threadIdx.y;
    int k       = blockIdx.x * TILE_X + threadIdx.x;
    int iIndex  = (i * iDim[1] * iDim[2]) + (j * iDim[2]) + k;
    if (i < iDim[0] && j < iDim[1] && k < iDim[2]) {
        int threads[3] = {threadIdx.z, threadIdx.y, threadIdx.x};
        buffer[threadIdx.z][threadIdx.y][threadIdx.x] = d_input[iIndex];
    }
    __syncthreads();

    if (i < iDim[0] && j < iDim[1] && k < iDim[2]) {
        d_output[iIndex] = buffer[threadIdx.z][threadIdx.y][threadIdx.x];
    }
}

__global__ void transpose_naive_kernel(const float* d_input, float* d_output,
                                       int dim0, int dim1, int dim2, int p0,
                                       int p1, int p2) {
    int idx[3] = {
        blockIdx.z * blockDim.z + threadIdx.z,  // i
        blockIdx.y * blockDim.y + threadIdx.y,  // j
        blockIdx.x * blockDim.x + threadIdx.x,  // k
    };
    if (idx[0] < dim0 && idx[1] < dim1 && idx[2] < dim2) {
        int iDim[3] = {dim0, dim1, dim2};
        int oDim[3] = {iDim[p0], iDim[p1], iDim[p2]};
        int odx[3]  = {idx[p0], idx[p1], idx[p2]};
        int iIndex = (idx[0] * iDim[1] * iDim[2]) + (idx[1] * iDim[2]) + idx[2];
        int oIndex = (odx[0] * oDim[1] * oDim[2]) + (odx[1] * oDim[2]) + odx[2];
        d_output[oIndex] = d_input[iIndex];
    }
}

__global__ void transpose_shmem_kernel(const float* d_input, float* d_output,
                                       int dim0, int dim1, int dim2, int p0,
                                       int p1, int p2) {
    __shared__ float buffer[TILE_Z][TILE_Y][TILE_X];

    int iDim[3] = {dim0, dim1, dim2};
    int i       = blockIdx.z * TILE_Z + threadIdx.z;
    int j       = blockIdx.y * TILE_Y + threadIdx.y;
    int k       = blockIdx.x * TILE_X + threadIdx.x;
    if (i < iDim[0] && j < iDim[1] && k < iDim[2]) {
        int iIndex     = (i * iDim[1] * iDim[2]) + (j * iDim[2]) + k;
        int threads[3] = {threadIdx.z, threadIdx.y, threadIdx.x};
        buffer[threads[p0]][threads[p1]][threads[p2]] = d_input[iIndex];
    }
    __syncthreads();

    int oDim[3]   = {iDim[p0], iDim[p1], iDim[p2]};
    int blocks[3] = {blockIdx.z, blockIdx.y, blockIdx.x};
    i             = blocks[p0] * TILE_Z + threadIdx.z;
    j             = blocks[p1] * TILE_Y + threadIdx.y;
    k             = blocks[p2] * TILE_X + threadIdx.x;
    if (i < oDim[0] && j < oDim[1] && k < oDim[2]) {
        int oIndex       = (i * oDim[1] * oDim[2]) + (j * oDim[2]) + k;
        d_output[oIndex] = buffer[threadIdx.z][threadIdx.y][threadIdx.x];
    }
}

__global__ void transpose_shmem_bank_kernel(const float* d_input,
                                            float* d_output, int dim0, int dim1,
                                            int dim2, int p0, int p1, int p2) {
    __shared__ float buffer[TILE_Z][TILE_Y][TILE_X + 1];

    int iDim[3] = {dim0, dim1, dim2};
    int i       = blockIdx.z * TILE_Z + threadIdx.z;
    int j       = blockIdx.y * TILE_Y + threadIdx.y;
    int k       = blockIdx.x * TILE_X + threadIdx.x;
    if (i < iDim[0] && j < iDim[1] && k < iDim[2]) {
        int iIndex     = (i * iDim[1] * iDim[2]) + (j * iDim[2]) + k;
        int threads[3] = {threadIdx.z, threadIdx.y, threadIdx.x};
        buffer[threads[p0]][threads[p1]][threads[p2]] = d_input[iIndex];
    }
    __syncthreads();

    int oDim[3]   = {iDim[p0], iDim[p1], iDim[p2]};
    int blocks[3] = {blockIdx.z, blockIdx.y, blockIdx.x};
    i             = blocks[p0] * TILE_Z + threadIdx.z;
    j             = blocks[p1] * TILE_Y + threadIdx.y;
    k             = blocks[p2] * TILE_X + threadIdx.x;
    if (i < oDim[0] && j < oDim[1] && k < oDim[2]) {
        int oIndex       = (i * oDim[1] * oDim[2]) + (j * oDim[2]) + k;
        d_output[oIndex] = buffer[threadIdx.z][threadIdx.y][threadIdx.x];
    }
}

__global__ void matrix_transpose_naive_kernel(const float* d_input,
                                              float* d_output, int m, int n) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < n && row < m) {
        d_output[col * m + row] = d_input[row * n + col];
    }
}

__global__ void matrix_transpose_kernel(const float* d_input, float* d_output,
                                        int dim0, int dim1) {
    __shared__ float buffer[TILE_Y][TILE_X + 1];

    // read matrix in linear order
    int j = blockIdx.x * TILE_X + threadIdx.x;
    int i = blockIdx.y * TILE_Y + threadIdx.y;
    if ((j < dim1) && (i < dim0)) {
        buffer[threadIdx.y][threadIdx.x] = d_input[i * dim1 + j];
    }
    __syncthreads();

    // write transposed matrix in linear order
    j = blockIdx.y * TILE_X + threadIdx.x;
    i = blockIdx.x * TILE_Y + threadIdx.y;
    if ((j < dim0) && (i < dim1)) {
        // transpose is done with buffer
        d_output[i * dim0 + j] = buffer[threadIdx.x][threadIdx.y];
    }
}


__global__ void transpose_021_kernel(const float* d_input, float* d_output,
                                     int dim0, int dim1, int dim2) {
    __shared__ float buffer[TILE_Y][TILE_X + 1];

    for (int k = 0; k < dim2; ++k) {
        int j = blockIdx.x * TILE_X + threadIdx.x;
        int i = blockIdx.y * TILE_Y + threadIdx.y;
        if ((j < dim1) && (i < dim0)) {
            int iIndex                       = k * dim0 * dim1 + i * dim1 + j;
            buffer[threadIdx.y][threadIdx.x] = d_input[iIndex];
        }
        __syncthreads();

        j = blockIdx.y * TILE_X + threadIdx.x;
        i = blockIdx.x * TILE_Y + threadIdx.y;
        if ((j < dim0) && (i < dim1)) {
            int oIndex       = k * dim0 * dim1 + i * dim0 + j;
            d_output[oIndex] = buffer[threadIdx.x][threadIdx.y];
        }
    }
}

__global__ void transpose_021_v2_kernel(const float* d_input, float* d_output,
                                        int dim0, int dim1, int dim2, int p0,
                                        int p1, int p2) {
    __shared__ float buffer[TILE_Z][TILE_Y][TILE_X + 1];

    int iDim[3] = {dim0, dim1, dim2};
    int i       = blockIdx.z * TILE_Z + threadIdx.z;
    int j       = blockIdx.y * TILE_Y + threadIdx.y;
    int k       = blockIdx.x * TILE_X + threadIdx.x;
    if (i < iDim[0] && j < iDim[1] && k < iDim[2]) {
        int iIndex     = (i * iDim[1] * iDim[2]) + (j * iDim[2]) + k;
        int threads[3] = {threadIdx.z, threadIdx.y, threadIdx.x};
        buffer[threads[p0]][threads[p1]][threads[p2]] = d_input[iIndex];
    }
    __syncthreads();

    int oDim[3]   = {iDim[p0], iDim[p1], iDim[p2]};
    int blocks[3] = {blockIdx.z, blockIdx.y, blockIdx.x};
    i             = blocks[p0] * TILE_Z + threadIdx.z;
    j             = blocks[p1] * TILE_Y + threadIdx.y;
    k             = blocks[p2] * TILE_X + threadIdx.x;
    if (i < oDim[0] && j < oDim[1] && k < oDim[2]) {
        int oIndex       = (i * oDim[1] * oDim[2]) + (j * oDim[2]) + k;
        d_output[oIndex] = buffer[threadIdx.z][threadIdx.y][threadIdx.x];
    }
}

// ============================================================================
// C++ SECTION
void array_rand_init(float* array, int size) {
    unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
    std::default_random_engine            generator(seed);
    std::uniform_real_distribution<float> distribution(0, 1);
    for (int i = 0; i < size; i++)
        array[i] = distribution(generator);
}

void array_print(const float* tensor, int size) {
    for (int i = 0; i < size; ++i) {
        std::cout << tensor[i] << ", ";
    }
    std::cout << std::endl;
}

void transpose_cpu(const float* input, float* output, const int* iDim,
                   const int* p) {
    int oDim[] = {iDim[p[0]], iDim[p[1]], iDim[p[2]]};
    for (int i = 0; i < iDim[0]; ++i) {
        for (int j = 0; j < iDim[1]; ++j) {
            for (int k = 0; k < iDim[2]; ++k) {
                int idx[] = {i, j, k};
                int odx[] = {idx[p[0]], idx[p[1]], idx[p[2]]};
                int iIndex =
                    (idx[0] * iDim[1] * iDim[2]) + (idx[1] * iDim[2]) + idx[2];
                int oIndex =
                    (odx[0] * oDim[1] * oDim[2]) + (odx[1] * oDim[2]) + odx[2];
                output[oIndex] = input[iIndex];
            }
        }
    }
}

void array_check(const float* gold, const float* result, int size) {
    // Check Correctness
    for (int i = 0; i < size; ++i) {
        if (result[i] != gold[i]) {
            std::cout << "!!!!!!!!!!!!! ERROR !!!!!!!!!!!!!" << std::endl;
            std::cout << "       i  = " << i << std::endl;
            std::cout << "  gold[i] = " << gold[i] << std::endl;
            std::cout << "result[i] = " << result[i] << std::endl;
            return;
        }
    }
}

void process(const float* gold, const float* result, int size, float host_ms,
             float kernel_ms, int bytes) {
    array_check(gold, result, size);
    kernel_ms /= NUM_REPS;
    std::cout << "       Time (ms): " << kernel_ms << std::endl;
    std::cout << " Bandwidth(GB/s): " << 2 * bytes * 1e-6 / kernel_ms
              << std::endl;
    std::cout << "    Speedup (ms): " << host_ms / kernel_ms << "x"
              << std::endl;

    if (ENABLE_PRINT) {
        array_print(gold, size);
        array_print(result, size);
    }
}

// ============================================================================
// MAIN
int main(int argc, char* argv[]) {
    // ------------------------------------------------------------------------
    // GET INFO
    int        size  = DIMENSION[0] * DIMENSION[1] * DIMENSION[2];
    const int  bytes = size * sizeof(float);
    const dim3 DimBlock(BLOCK_X, BLOCK_Y, BLOCK_Z);
    const dim3 DimGrid(std::ceil((float)DIMENSION[2] / DimBlock.x),
                       std::ceil((float)DIMENSION[1] / DimBlock.y),
                       std::ceil((float)DIMENSION[0] / DimBlock.z));
    // ------------------------------------------------------------------------
    // PRINT INFO
    std::cout << "TILE:    " << T << std::endl;
    std::cout << "size:    " << size << std::endl;
    std::cout << "DIMENSION:   (" << DIMENSION[0] << ", " << DIMENSION[1]
              << ", " << DIMENSION[2] << ")" << std::endl;
    std::cout << "PERMUTATION: (" << PERMUTATION[0] << ", " << PERMUTATION[1]
              << ", " << PERMUTATION[2] << ")" << std::endl;
    std::cout << "DimBlock:    (" << DimBlock.x << ", " << DimBlock.y << ", "
              << DimBlock.z << ")" << std::endl;
    std::cout << "DimGrid:     (" << DimGrid.x << ", " << DimGrid.y << ", "
              << DimGrid.z << ")" << std::endl;
    std::cout << std::endl;


    // ------------------------------------------------------------------------
    // SETUP TIMERS
    float       host_ms, kernel_ms;
    hipEvent_t startEvent, stopEvent;
    CHECK_CUDA(hipEventCreate(&startEvent));
    CHECK_CUDA(hipEventCreate(&stopEvent));
    std::chrono::steady_clock::time_point start;
    std::chrono::steady_clock::time_point end;


    // ------------------------------------------------------------------------
    // HOST MEMORY ALLOCATION AND INITIALIZATION
    float* h_input  = new float[size]{};
    float* h_output = new float[size]{};
    float* h_gold   = new float[size]{};

    for (int i = 0; i < size; ++i)
        h_input[i] = i * 1.0f;
    // array_rand_init(h_input, size);


    // ------------------------------------------------------------------------
    // HOST EXECUTION
    start = std::chrono::steady_clock::now();
    for (int i = 0; i < NUM_REPS; ++i)
        transpose_cpu(h_input, h_gold, DIMENSION, PERMUTATION);
    end     = std::chrono::steady_clock::now();
    host_ms = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start)
                  .count() *
              1e-6 / NUM_REPS;
    std::cout << "           Host Time (ms): " << host_ms << std::endl;


    // ------------------------------------------------------------------------
    // DEVICE MEMORY ALLOCATION AND INITIALIZATION
    float *d_input, *d_output;
    CHECK_CUDA(hipMalloc(&d_input, bytes));
    CHECK_CUDA(hipMalloc(&d_output, bytes));

    CHECK_CUDA(hipMemcpy(d_input, h_input, bytes, hipMemcpyHostToDevice));


    // ------------------------------------------------------------------------
    // GENERARE COPY BANDWITH
    CHECK_CUDA(hipMemset(d_output, 0, bytes));  // Initialize output
    copy_kernel<<<DimGrid, DimBlock>>>(
        d_input, d_output, DIMENSION[0], DIMENSION[1], DIMENSION[2],
        PERMUTATION[0], PERMUTATION[1], PERMUTATION[2]);  // warmup
    CHECK_CUDA(hipEventRecord(startEvent, 0));
    for (int i = 0; i < NUM_REPS; ++i)
        copy_kernel<<<DimGrid, DimBlock>>>(
            d_input, d_output, DIMENSION[0], DIMENSION[1], DIMENSION[2],
            PERMUTATION[0], PERMUTATION[1], PERMUTATION[2]);
    CHECK_CUDA(hipEventRecord(stopEvent, 0));
    CHECK_CUDA(hipEventSynchronize(stopEvent));
    CHECK_CUDA(hipEventElapsedTime(&kernel_ms, startEvent, stopEvent));
    CHECK_CUDA(hipMemcpy(h_output, d_output, bytes, hipMemcpyDeviceToHost));
    array_check(h_input, h_output, size);
    kernel_ms /= NUM_REPS;
    std::cout << "    Copy Bandwidth (GB/s): " << 2 * bytes * 1e-6 / kernel_ms
              << std::endl;


    // ------------------------------------------------------------------------
    // GENERARE COPY BANDWITH
    CHECK_CUDA(hipMemset(d_output, 0, bytes));  // Initialize output
    copy_shm_kernel<<<DimGrid, DimBlock>>>(
        d_input, d_output, DIMENSION[0], DIMENSION[1], DIMENSION[2],
        PERMUTATION[0], PERMUTATION[1], PERMUTATION[2]);  // warmup
    CHECK_CUDA(hipEventRecord(startEvent, 0));
    for (int i = 0; i < NUM_REPS; ++i)
        copy_shm_kernel<<<DimGrid, DimBlock>>>(
            d_input, d_output, DIMENSION[0], DIMENSION[1], DIMENSION[2],
            PERMUTATION[0], PERMUTATION[1], PERMUTATION[2]);
    CHECK_CUDA(hipEventRecord(stopEvent, 0));
    CHECK_CUDA(hipEventSynchronize(stopEvent));
    CHECK_CUDA(hipEventElapsedTime(&kernel_ms, startEvent, stopEvent));
    CHECK_CUDA(hipMemcpy(h_output, d_output, bytes, hipMemcpyDeviceToHost));
    array_check(h_input, h_output, size);
    kernel_ms /= NUM_REPS;
    std::cout << "Copy Bandwidth SHM (GB/s): " << 2 * bytes * 1e-6 / kernel_ms
              << std::endl;


    // ------------------------------------------------------------------------
    // TRANSPOSE NAIVE
    std::cout << "\nTranspose Naive" << std::endl;
    CHECK_CUDA(hipMemset(d_output, 0, bytes));  // Initialize output
    transpose_naive_kernel<<<DimGrid, DimBlock>>>(
        d_input, d_output, DIMENSION[0], DIMENSION[1], DIMENSION[2],
        PERMUTATION[0], PERMUTATION[1], PERMUTATION[2]);  // warmup
    CHECK_CUDA(hipEventRecord(startEvent, 0));
    for (int i = 0; i < NUM_REPS; ++i)
        transpose_naive_kernel<<<DimGrid, DimBlock>>>(
            d_input, d_output, DIMENSION[0], DIMENSION[1], DIMENSION[2],
            PERMUTATION[0], PERMUTATION[1], PERMUTATION[2]);
    CHECK_CUDA(hipEventRecord(stopEvent, 0));
    CHECK_CUDA(hipEventSynchronize(stopEvent));
    CHECK_CUDA(hipEventElapsedTime(&kernel_ms, startEvent, stopEvent));
    CHECK_CUDA(hipMemcpy(h_output, d_output, bytes, hipMemcpyDeviceToHost));
    process(h_gold, h_output, size, host_ms, kernel_ms, bytes);


    // ------------------------------------------------------------------------
    // TRANSPOSE WITH SHARED MEMORY
    std::cout << "\nTranspose with shared Memory" << std::endl;
    CHECK_CUDA(hipMemset(d_output, 0, bytes));  // Initialize output
    transpose_shmem_kernel<<<DimGrid, DimBlock>>>(
        d_input, d_output, DIMENSION[0], DIMENSION[1], DIMENSION[2],
        PERMUTATION[0], PERMUTATION[1], PERMUTATION[2]);  // warmup
    CHECK_CUDA(hipEventRecord(startEvent, 0));
    for (int i = 0; i < NUM_REPS; ++i)
        transpose_shmem_kernel<<<DimGrid, DimBlock>>>(
            d_input, d_output, DIMENSION[0], DIMENSION[1], DIMENSION[2],
            PERMUTATION[0], PERMUTATION[1], PERMUTATION[2]);
    CHECK_CUDA(hipEventRecord(stopEvent, 0));
    CHECK_CUDA(hipEventSynchronize(stopEvent));
    CHECK_CUDA(hipEventElapsedTime(&kernel_ms, startEvent, stopEvent));
    CHECK_CUDA(hipMemcpy(h_output, d_output, bytes, hipMemcpyDeviceToHost));
    process(h_gold, h_output, size, host_ms, kernel_ms, bytes);


    // ------------------------------------------------------------------------
    // TRANSPOSE WITH SHARED MEMORY AND BANK CONFLICT AVOIDANCE
    std::cout << "\nTranspose with shared Memory and Bank Conflict avoidance"
              << std::endl;
    CHECK_CUDA(hipMemset(d_output, 0, bytes));  // Initialize output
    transpose_shmem_bank_kernel<<<DimGrid, DimBlock>>>(
        d_input, d_output, DIMENSION[0], DIMENSION[1], DIMENSION[2],
        PERMUTATION[0], PERMUTATION[1], PERMUTATION[2]);  // warmup
    CHECK_CUDA(hipEventRecord(startEvent, 0));
    for (int i = 0; i < NUM_REPS; ++i)
        transpose_shmem_bank_kernel<<<DimGrid, DimBlock>>>(
            d_input, d_output, DIMENSION[0], DIMENSION[1], DIMENSION[2],
            PERMUTATION[0], PERMUTATION[1], PERMUTATION[2]);
    CHECK_CUDA(hipEventRecord(stopEvent, 0));
    CHECK_CUDA(hipEventSynchronize(stopEvent));
    CHECK_CUDA(hipEventElapsedTime(&kernel_ms, startEvent, stopEvent));
    CHECK_CUDA(hipMemcpy(h_output, d_output, bytes, hipMemcpyDeviceToHost));
    process(h_gold, h_output, size, host_ms, kernel_ms, bytes);


    // ------------------------------------------------------------------------
    // MATRIX TRANSPOSE 021
    std::cout << "\nMatrix Transpose 021" << std::endl;
    const dim3 DimBlockMatrix(BLOCK_X, BLOCK_Y, 1);
    const dim3 DimGridMatrix(std::ceil((float)DIMENSION[2] / DimBlock.x),
                             std::ceil((float)DIMENSION[1] / DimBlock.y), 1);
    CHECK_CUDA(hipMemset(d_output, 0, bytes));  // Initialize output
    transpose_021_kernel<<<DimGridMatrix, DimBlockMatrix>>>(
        d_input, d_output, DIMENSION[1], DIMENSION[2], DIMENSION[0]);  // warmup
    CHECK_CUDA(hipEventRecord(startEvent, 0));
    for (int i = 0; i < NUM_REPS; ++i)
        transpose_021_kernel<<<DimGridMatrix, DimBlockMatrix>>>(
            d_input, d_output, DIMENSION[1], DIMENSION[2], DIMENSION[0]);
    CHECK_CUDA(hipEventRecord(stopEvent, 0));
    CHECK_CUDA(hipEventSynchronize(stopEvent));
    CHECK_CUDA(hipEventElapsedTime(&kernel_ms, startEvent, stopEvent));
    CHECK_CUDA(hipMemcpy(h_output, d_output, bytes, hipMemcpyDeviceToHost));
    process(h_gold, h_output, size, host_ms, kernel_ms, bytes);


    // ------------------------------------------------------------------------
    // GENERARE COPY BANDWITH
    const dim3 DimBlock2(BLOCK_X, BLOCK_Y, 1);
    const dim3 DimGrid2(std::ceil((float)DIMENSION[2] / DimBlock.x),
                        std::ceil((float)DIMENSION[1] / DimBlock.y),
                        std::ceil((float)DIMENSION[0] / DimBlock.z));
    CHECK_CUDA(hipMemset(d_output, 0, bytes));  // Initialize output
    copy_shm_kernel<<<DimGrid2, DimBlock2>>>(
        d_input, d_output, DIMENSION[0], DIMENSION[1], DIMENSION[2],
        PERMUTATION[0], PERMUTATION[1], PERMUTATION[2]);  // warmup
    CHECK_CUDA(hipEventRecord(startEvent, 0));
    for (int i = 0; i < NUM_REPS; ++i)
        copy_shm_kernel<<<DimGrid, DimBlock>>>(
            d_input, d_output, DIMENSION[0], DIMENSION[1], DIMENSION[2],
            PERMUTATION[0], PERMUTATION[1], PERMUTATION[2]);
    CHECK_CUDA(hipEventRecord(stopEvent, 0));
    CHECK_CUDA(hipEventSynchronize(stopEvent));
    CHECK_CUDA(hipEventElapsedTime(&kernel_ms, startEvent, stopEvent));
    CHECK_CUDA(hipMemcpy(h_output, d_output, bytes, hipMemcpyDeviceToHost));
    array_check(h_input, h_output, size);
    kernel_ms /= NUM_REPS;
    std::cout << "Copy Bandwidth REKT (GB/s): " << 2 * bytes * 1e-6 / kernel_ms
              << std::endl;


    // ------------------------------------------------------------------------
    // MATRIX TRANSPOSE 021 v2
    std::cout << "\nMatrix Transpose REKT VERSION" << std::endl;
    const dim3 DimBlockMatrix2(16, 16, 1);
    const dim3 DimGridMatrix2(std::ceil((float)DIMENSION[2] / DimBlock.x),
                              std::ceil((float)DIMENSION[1] / DimBlock.y),
                              std::ceil((float)DIMENSION[0] / DimBlock.z));
    CHECK_CUDA(hipMemset(d_output, 0, bytes));  // Initialize output
    transpose_021_v2_kernel<<<DimGridMatrix2, DimBlockMatrix2>>>(
        d_input, d_output, DIMENSION[0], DIMENSION[1], DIMENSION[2],
        PERMUTATION[0], PERMUTATION[1], PERMUTATION[2]);  // warmup
    CHECK_CUDA(hipEventRecord(startEvent, 0));
    for (int i = 0; i < NUM_REPS; ++i)
        transpose_021_v2_kernel<<<DimGridMatrix, DimBlockMatrix>>>(
            d_input, d_output, DIMENSION[0], DIMENSION[1], DIMENSION[2],
            PERMUTATION[0], PERMUTATION[1], PERMUTATION[2]);
    CHECK_CUDA(hipEventRecord(stopEvent, 0));
    CHECK_CUDA(hipEventSynchronize(stopEvent));
    CHECK_CUDA(hipEventElapsedTime(&kernel_ms, startEvent, stopEvent));
    CHECK_CUDA(hipMemcpy(h_output, d_output, bytes, hipMemcpyDeviceToHost));
    process(h_gold, h_output, size, host_ms, kernel_ms, bytes);


    // ------------------------------------------------------------------------
    // MATRIX TRANSPOSE Naive
    // std::cout << "\nMatrix Transpose naive" << std::endl;
    // CHECK_CUDA(cudaMemset(d_output, 0, bytes));  // Initialize output
    // matrix_transpose_naive_kernel<<<DimGridMatrix, DimBlockMatrix>>>(
    //     d_input, d_output, DIMENSION[1], DIMENSION[2]);  // warmup
    // CHECK_CUDA(cudaEventRecord(startEvent, 0));
    // for (int i = 0; i < NUM_REPS; ++i)
    //     matrix_transpose_naive_kernel<<<DimGridMatrix, DimBlockMatrix>>>(
    //         d_input, d_output, DIMENSION[1], DIMENSION[2]);
    // CHECK_CUDA(cudaEventRecord(stopEvent, 0));
    // CHECK_CUDA(cudaEventSynchronize(stopEvent));
    // CHECK_CUDA(cudaEventElapsedTime(&kernel_ms, startEvent, stopEvent));
    // CHECK_CUDA(cudaMemcpy(h_output, d_output, bytes,
    // cudaMemcpyDeviceToHost)); process(h_gold, h_output, size, host_ms,
    // kernel_ms, bytes);


    // ------------------------------------------------------------------------
    // MATRIX TRANSPOSE
    // std::cout
    //     << "\nMatrix Transpose with shared Memory and Bank Conflict avoidance
    //     "
    //     << std::endl;
    // CHECK_CUDA(cudaMemset(d_output, 0, bytes));  // Initialize output
    // matrix_transpose_kernel<<<DimGridMatrix, DimBlockMatrix>>>(
    //     d_input, d_output, DIMENSION[1], DIMENSION[2]);  // warmup
    // CHECK_CUDA(cudaEventRecord(startEvent, 0));
    // for (int i = 0; i < NUM_REPS; ++i)
    //     matrix_transpose_kernel<<<DimGridMatrix, DimBlockMatrix>>>(
    //         d_input, d_output, DIMENSION[1], DIMENSION[2]);
    // CHECK_CUDA(cudaEventRecord(stopEvent, 0));
    // CHECK_CUDA(cudaEventSynchronize(stopEvent));
    // CHECK_CUDA(cudaEventElapsedTime(&kernel_ms, startEvent, stopEvent));
    // CHECK_CUDA(cudaMemcpy(h_output, d_output, bytes,
    // cudaMemcpyDeviceToHost)); process(h_gold, h_output, size, host_ms,
    // kernel_ms, bytes);


    // ------------------------------------------------------------------------
    // CLEAN SHUTDOWN
    delete[] h_input;
    delete[] h_output;
    delete[] h_gold;

    CHECK_CUDA(hipEventDestroy(startEvent));
    CHECK_CUDA(hipEventDestroy(stopEvent));
    CHECK_CUDA(hipFree(d_input));
    CHECK_CUDA(hipFree(d_output));
    hipDeviceReset();

    return 0;
}
