
#include <hip/hip_runtime.h>
#include <cassert>
#include <chrono>
#include <fstream>
#include <iostream>
#include <random>


// ============================================================================
// SETTINGS
const int  NUM_REPS     = 100;
const bool ENABLE_PRINT = false;
const int  TILE         = 8;


// ============================================================================
// CUDA SECTION
inline hipError_t CHECK_CUDA(hipError_t result) {
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
}

__global__ void copy_simple_kernel(const float* d_input, float* d_output,
                                   int dimz, int dimy, int dimx) {
    int x     = blockIdx.x * blockDim.x + threadIdx.x;
    int y     = blockIdx.y * blockDim.y + threadIdx.y;
    int z     = blockIdx.z * blockDim.z + threadIdx.z;
    int index = (z * dimy * dimx) + (y * dimx) + x;

    if (z < dimz && y < dimy && x < dimx) {
        d_output[index] = d_input[index];
    }
}

__global__ void copy_shm_kernel(const float* d_input, float* d_output, int dimz,
                                int dimy, int dimx) {
    __shared__ float buffer[TILE][TILE][TILE];

    int x     = blockIdx.x * TILE + threadIdx.x;
    int y     = blockIdx.y * TILE + threadIdx.y;
    int z     = blockIdx.z * TILE + threadIdx.z;
    int index = (z * dimy * dimx) + (y * dimx) + x;

    if (z < dimz && y < dimy && x < dimx) {
        buffer[threadIdx.z][threadIdx.y][threadIdx.x] = d_input[index];
    }
    __syncthreads();

    if (z < dimz && y < dimy && x < dimx) {
        d_output[index] = buffer[threadIdx.z][threadIdx.y][threadIdx.x];
    }
}

__global__ void transpose_simple_kernel(const float* d_input, float* d_output,
                                        int dimz, int dimy, int dimx, int pz,
                                        int py, int px) {
    int idx[3]  = {blockIdx.z * blockDim.z + threadIdx.z,
                  blockIdx.y * blockDim.y + threadIdx.y,
                  blockIdx.x * blockDim.x + threadIdx.x};
    int iDim[3] = {dimz, dimy, dimx};
    int oDim[3] = {iDim[pz], iDim[py], iDim[px]};
    int odx[3]  = {idx[pz], idx[py], idx[px]};
    int iIndex  = (idx[0] * iDim[1] * iDim[2]) + (idx[1] * iDim[2]) + idx[2];
    int oIndex  = (odx[0] * oDim[1] * oDim[2]) + (odx[1] * oDim[2]) + odx[2];

    if (idx[0] < dimz && idx[1] < dimy && idx[2] < dimx) {
        d_output[oIndex] = d_input[iIndex];
    }
}

__global__ void transpose_shm_kernel(const float* d_input, float* d_output,
                                     int dimz, int dimy, int dimx, int pz,
                                     int py, int px) {
    __shared__ float buffer[TILE][TILE][TILE];

    int iDim[3] = {dimz, dimy, dimx};
    int x       = blockIdx.x * TILE + threadIdx.x;
    int y       = blockIdx.y * TILE + threadIdx.y;
    int z       = blockIdx.z * TILE + threadIdx.z;
    if (z < iDim[0] && y < iDim[1] && x < iDim[2]) {
        int iIndex     = (z * iDim[1] * iDim[2]) + (y * iDim[2]) + x;
        int threads[3] = {threadIdx.z, threadIdx.y, threadIdx.x};
        buffer[threads[pz]][threads[py]][threads[px]] = d_input[iIndex];
    }
    __syncthreads();

    int oDim[3]   = {iDim[pz], iDim[py], iDim[px]};
    int blocks[3] = {blockIdx.z, blockIdx.y, blockIdx.x};
    x             = blocks[px] * TILE + threadIdx.x;
    y             = blocks[py] * TILE + threadIdx.y;
    z             = blocks[pz] * TILE + threadIdx.z;
    if (z < oDim[0] && y < oDim[1] && x < oDim[2]) {
        int oIndex       = (z * oDim[1] * oDim[2]) + (y * oDim[2]) + x;
        d_output[oIndex] = buffer[threadIdx.z][threadIdx.y][threadIdx.x];
    }
}

__global__ void transpose_shm_bank_kernel(const float* d_input, float* d_output,
                                          int dimz, int dimy, int dimx, int pz,
                                          int py, int px) {
    __shared__ float buffer[TILE][TILE][TILE + 1];

    int iDim[3] = {dimz, dimy, dimx};
    int x       = blockIdx.x * TILE + threadIdx.x;
    int y       = blockIdx.y * TILE + threadIdx.y;
    int z       = blockIdx.z * TILE + threadIdx.z;
    if (z < iDim[0] && y < iDim[1] && x < iDim[2]) {
        int iIndex     = (z * iDim[1] * iDim[2]) + (y * iDim[2]) + x;
        int threads[3] = {threadIdx.z, threadIdx.y, threadIdx.x};
        buffer[threads[pz]][threads[py]][threads[px]] = d_input[iIndex];
    }
    __syncthreads();

    int oDim[3]   = {iDim[pz], iDim[py], iDim[px]};
    int blocks[3] = {blockIdx.z, blockIdx.y, blockIdx.x};
    x             = blocks[px] * TILE + threadIdx.x;
    y             = blocks[py] * TILE + threadIdx.y;
    z             = blocks[pz] * TILE + threadIdx.z;
    if (z < oDim[0] && y < oDim[1] && x < oDim[2]) {
        int oIndex       = (z * oDim[1] * oDim[2]) + (y * oDim[2]) + x;
        d_output[oIndex] = buffer[threadIdx.z][threadIdx.y][threadIdx.x];
    }
}

__global__ void transpose_simple_012_kernel(const float* d_input,
                                            float* d_output, int dimz, int dimy,
                                            int dimx) {
    int x      = blockIdx.x * blockDim.x + threadIdx.x;
    int y      = blockIdx.y * blockDim.y + threadIdx.y;
    int z      = blockIdx.z * blockDim.z + threadIdx.z;
    int iIndex = (z * dimy * dimx) + (y * dimx) + x;
    int oIndex = (z * dimy * dimx) + (y * dimx) + x;

    if (z < dimz && y < dimy && x < dimx) {
        d_output[oIndex] = d_input[iIndex];
    }
}

__global__ void transpose_simple_021_kernel(const float* d_input,
                                            float* d_output, int dimz, int dimy,
                                            int dimx) {
    int x      = blockIdx.x * blockDim.x + threadIdx.x;
    int y      = blockIdx.y * blockDim.y + threadIdx.y;
    int z      = blockIdx.z * blockDim.z + threadIdx.z;
    int iIndex = (z * dimy * dimx) + (y * dimx) + x;
    int oIndex = (z * dimx * dimy) + (x * dimy) + y;

    if (z < dimz && y < dimy && x < dimx) {
        d_output[oIndex] = d_input[iIndex];
    }
}

__global__ void transpose_simple_102_kernel(const float* d_input,
                                            float* d_output, int dimz, int dimy,
                                            int dimx) {
    int x      = blockIdx.x * blockDim.x + threadIdx.x;
    int y      = blockIdx.y * blockDim.y + threadIdx.y;
    int z      = blockIdx.z * blockDim.z + threadIdx.z;
    int iIndex = (z * dimy * dimx) + (y * dimx) + x;
    int oIndex = (y * dimz * dimx) + (z * dimx) + x;

    if (z < dimz && y < dimy && x < dimx) {
        d_output[oIndex] = d_input[iIndex];
    }
}

__global__ void transpose_simple_120_kernel(const float* d_input,
                                            float* d_output, int dimz, int dimy,
                                            int dimx) {
    int x      = blockIdx.x * blockDim.x + threadIdx.x;
    int y      = blockIdx.y * blockDim.y + threadIdx.y;
    int z      = blockIdx.z * blockDim.z + threadIdx.z;
    int iIndex = (z * dimy * dimx) + (y * dimx) + x;
    int oIndex = (y * dimx * dimz) + (x * dimz) + z;

    if (z < dimz && y < dimy && x < dimx) {
        d_output[oIndex] = d_input[iIndex];
    }
}

__global__ void transpose_simple_201_kernel(const float* d_input,
                                            float* d_output, int dimz, int dimy,
                                            int dimx) {
    int x      = blockIdx.x * blockDim.x + threadIdx.x;
    int y      = blockIdx.y * blockDim.y + threadIdx.y;
    int z      = blockIdx.z * blockDim.z + threadIdx.z;
    int iIndex = (z * dimy * dimx) + (y * dimx) + x;
    int oIndex = (x * dimz * dimy) + (z * dimy) + y;

    if (z < dimz && y < dimy && x < dimx) {
        d_output[oIndex] = d_input[iIndex];
    }
}

__global__ void transpose_simple_210_kernel(const float* d_input,
                                            float* d_output, int dimz, int dimy,
                                            int dimx) {
    int x      = blockIdx.x * blockDim.x + threadIdx.x;
    int y      = blockIdx.y * blockDim.y + threadIdx.y;
    int z      = blockIdx.z * blockDim.z + threadIdx.z;
    int iIndex = (z * dimy * dimx) + (y * dimx) + x;
    int oIndex = (x * dimy * dimz) + (y * dimz) + z;

    if (z < dimz && y < dimy && x < dimx) {
        d_output[oIndex] = d_input[iIndex];
    }
}


void transpose_simple_selector(const dim3& DimGrid, const dim3& DimBlock,
                               const float* d_input, float* d_output,
                               const int* dim, const int* perm) {
    if (perm[0] == 0 && perm[1] == 1 && perm[2] == 2) {
        transpose_simple_012_kernel<<<DimGrid, DimBlock>>>(
            d_input, d_output, dim[0], dim[1], dim[2]);
    } else if (perm[0] == 0 && perm[1] == 2 && perm[2] == 1) {
        transpose_simple_021_kernel<<<DimGrid, DimBlock>>>(
            d_input, d_output, dim[0], dim[1], dim[2]);
    } else if (perm[0] == 1 && perm[1] == 0 && perm[2] == 2) {
        transpose_simple_102_kernel<<<DimGrid, DimBlock>>>(
            d_input, d_output, dim[0], dim[1], dim[2]);
    } else if (perm[0] == 1 && perm[1] == 2 && perm[2] == 0) {
        transpose_simple_120_kernel<<<DimGrid, DimBlock>>>(
            d_input, d_output, dim[0], dim[1], dim[2]);
    } else if (perm[0] == 2 && perm[1] == 0 && perm[2] == 1) {
        transpose_simple_201_kernel<<<DimGrid, DimBlock>>>(
            d_input, d_output, dim[0], dim[1], dim[2]);
    } else if (perm[0] == 2 && perm[1] == 1 && perm[2] == 0) {
        transpose_simple_210_kernel<<<DimGrid, DimBlock>>>(
            d_input, d_output, dim[0], dim[1], dim[2]);
    }
}

__global__ void transpose_shm_012_kernel(const float* d_input, float* d_output,
                                         int dimz, int dimy, int dimx) {
    __shared__ float buffer[TILE][TILE][TILE];

    int x = blockIdx.x * TILE + threadIdx.x;
    int y = blockIdx.y * TILE + threadIdx.y;
    int z = blockIdx.z * TILE + threadIdx.z;
    if (z < dimz && y < dimy && x < dimx) {
        int iIndex = (z * dimy * dimx) + (y * dimx) + x;
        buffer[threadIdx.z][threadIdx.y][threadIdx.x] = d_input[iIndex];
    }
    __syncthreads();

    if (z < dimz && y < dimy && x < dimx) {
        int oIndex       = (z * dimy * dimx) + (y * dimx) + x;
        d_output[oIndex] = buffer[threadIdx.z][threadIdx.y][threadIdx.x];
    }
}


__global__ void transpose_shm_021_kernel(const float* d_input, float* d_output,
                                         int dimz, int dimy, int dimx) {
    __shared__ float buffer[TILE][TILE][TILE];

    int x = blockIdx.x * TILE + threadIdx.x;
    int y = blockIdx.y * TILE + threadIdx.y;
    int z = blockIdx.z * TILE + threadIdx.z;
    if (z < dimz && y < dimy && x < dimx) {
        int iIndex = (z * dimy * dimx) + (y * dimx) + x;
        buffer[threadIdx.z][threadIdx.x][threadIdx.y] = d_input[iIndex];
    }
    __syncthreads();

    if (z < dimz && y < dimx && x < dimy) {
        int oIndex       = (z * dimx * dimy) + (x * dimy) + y;
        d_output[oIndex] = buffer[threadIdx.z][threadIdx.y][threadIdx.x];
    }
}


__global__ void transpose_shm_102_kernel(const float* d_input, float* d_output,
                                         int dimz, int dimy, int dimx) {
    __shared__ float buffer[TILE][TILE][TILE];

    int x = blockIdx.x * TILE + threadIdx.x;
    int y = blockIdx.y * TILE + threadIdx.y;
    int z = blockIdx.z * TILE + threadIdx.z;
    if (z < dimz && y < dimy && x < dimx) {
        int iIndex = (z * dimy * dimx) + (y * dimx) + x;
        buffer[threadIdx.y][threadIdx.z][threadIdx.x] = d_input[iIndex];
    }
    __syncthreads();

    if (z < dimy && y < dimz && x < dimx) {
        int oIndex       = (y * dimz * dimx) + (z * dimx) + x;
        d_output[oIndex] = buffer[threadIdx.z][threadIdx.y][threadIdx.x];
    }
}


__global__ void transpose_shm_120_kernel(const float* d_input, float* d_output,
                                         int dimz, int dimy, int dimx) {
    __shared__ float buffer[TILE][TILE][TILE];

    int x = blockIdx.x * TILE + threadIdx.x;
    int y = blockIdx.y * TILE + threadIdx.y;
    int z = blockIdx.z * TILE + threadIdx.z;
    if (z < dimz && y < dimy && x < dimx) {
        int iIndex = (z * dimy * dimx) + (y * dimx) + x;
        buffer[threadIdx.y][threadIdx.x][threadIdx.z] = d_input[iIndex];
    }
    __syncthreads();

    x = blockIdx.z * TILE + threadIdx.x;
    y = blockIdx.x * TILE + threadIdx.y;
    z = blockIdx.y * TILE + threadIdx.z;
    if (z < dimy && y < dimx && x < dimz) {
        int oIndex       = (z * dimx * dimz) + (y * dimz) + x;
        d_output[oIndex] = buffer[threadIdx.z][threadIdx.y][threadIdx.x];
    }
}


__global__ void transpose_shm_201_kernel(const float* d_input, float* d_output,
                                         int dimz, int dimy, int dimx) {
    __shared__ float buffer[TILE][TILE][TILE];

    int x = blockIdx.x * TILE + threadIdx.x;
    int y = blockIdx.y * TILE + threadIdx.y;
    int z = blockIdx.z * TILE + threadIdx.z;
    if (z < dimz && y < dimy && x < dimx) {
        int iIndex = (x * dimy * dimx) + (z * dimx) + y;
        buffer[threadIdx.x][threadIdx.z][threadIdx.y] = d_input[iIndex];
    }
    __syncthreads();

    if (z < dimx && y < dimz && x < dimy) {
        int oIndex       = (z * dimz * dimy) + (y * dimy) + x;
        d_output[oIndex] = buffer[threadIdx.z][threadIdx.y][threadIdx.x];
    }
}


__global__ void transpose_shm_210_kernel(const float* d_input, float* d_output,
                                         int dimz, int dimy, int dimx) {
    __shared__ float buffer[TILE][TILE][TILE];

    int x = blockIdx.x * TILE + threadIdx.x;
    int y = blockIdx.y * TILE + threadIdx.y;
    int z = blockIdx.z * TILE + threadIdx.z;
    if (z < dimz && y < dimy && x < dimx) {
        int iIndex = (z * dimy * dimx) + (y * dimx) + x;
        buffer[threadIdx.x][threadIdx.y][threadIdx.z] = d_input[iIndex];
    }
    __syncthreads();

    if (z < dimx && y < dimy && x < dimz) {
        int oIndex       = (x * dimy * dimz) + (y * dimz) + z;
        d_output[oIndex] = buffer[threadIdx.z][threadIdx.y][threadIdx.x];
    }
}

void transpose_shm_selector(const dim3& DimGrid, const dim3& DimBlock,
                            const float* d_input, float* d_output,
                            const int* dim, const int* perm) {
    if (perm[0] == 0 && perm[1] == 1 && perm[2] == 2) {
        transpose_shm_012_kernel<<<DimGrid, DimBlock>>>(d_input, d_output,
                                                        dim[0], dim[1], dim[2]);
    } else if (perm[0] == 0 && perm[1] == 2 && perm[2] == 1) {
        transpose_shm_021_kernel<<<DimGrid, DimBlock>>>(d_input, d_output,
                                                        dim[0], dim[1], dim[2]);
    } else if (perm[0] == 1 && perm[1] == 0 && perm[2] == 2) {
        transpose_shm_102_kernel<<<DimGrid, DimBlock>>>(d_input, d_output,
                                                        dim[0], dim[1], dim[2]);
    } else if (perm[0] == 1 && perm[1] == 2 && perm[2] == 0) {
        transpose_shm_120_kernel<<<DimGrid, DimBlock>>>(d_input, d_output,
                                                        dim[0], dim[1], dim[2]);
    } else if (perm[0] == 2 && perm[1] == 0 && perm[2] == 1) {
        transpose_shm_201_kernel<<<DimGrid, DimBlock>>>(d_input, d_output,
                                                        dim[0], dim[1], dim[2]);
    } else if (perm[0] == 2 && perm[1] == 1 && perm[2] == 0) {
        transpose_shm_210_kernel<<<DimGrid, DimBlock>>>(d_input, d_output,
                                                        dim[0], dim[1], dim[2]);
    }
}

template<int pz, int py, int px>
__global__ void transpose_shm_kernel_tmpl(const float* d_input, float* d_output,
                                          int dimz, int dimy, int dimx) {
    __shared__ float buffer[TILE][TILE][TILE];

    int iDim[3] = {dimz, dimy, dimx};
    int x       = blockIdx.x * TILE + threadIdx.x;
    int y       = blockIdx.y * TILE + threadIdx.y;
    int z       = blockIdx.z * TILE + threadIdx.z;
    if (z < iDim[0] && y < iDim[1] && x < iDim[2]) {
        int iIndex     = (z * iDim[1] * iDim[2]) + (y * iDim[2]) + x;
        int threads[3] = {threadIdx.z, threadIdx.y, threadIdx.x};
        buffer[threads[pz]][threads[py]][threads[px]] = d_input[iIndex];
    }
    __syncthreads();

    int oDim[3]   = {iDim[pz], iDim[py], iDim[px]};
    int blocks[3] = {blockIdx.z, blockIdx.y, blockIdx.x};
    x             = blocks[px] * TILE + threadIdx.x;
    y             = blocks[py] * TILE + threadIdx.y;
    z             = blocks[pz] * TILE + threadIdx.z;
    if (z < oDim[0] && y < oDim[1] && x < oDim[2]) {
        int oIndex       = (z * oDim[1] * oDim[2]) + (y * oDim[2]) + x;
        d_output[oIndex] = buffer[threadIdx.z][threadIdx.y][threadIdx.x];
    }
}

void transpose_shm_tmpl_selector(const dim3& DimGrid, const dim3& DimBlock,
                                 const float* d_input, float* d_output,
                                 const int* dim, const int* perm) {
    if (perm[0] == 0 && perm[1] == 1 && perm[2] == 2) {
        transpose_shm_kernel_tmpl<0, 1, 2>
            <<<DimGrid, DimBlock>>>(d_input, d_output, dim[0], dim[1], dim[2]);
    } else if (perm[0] == 0 && perm[1] == 2 && perm[2] == 1) {
        transpose_shm_kernel_tmpl<0, 2, 1>
            <<<DimGrid, DimBlock>>>(d_input, d_output, dim[0], dim[1], dim[2]);
    } else if (perm[0] == 1 && perm[1] == 0 && perm[2] == 2) {
        transpose_shm_kernel_tmpl<1, 0, 2>
            <<<DimGrid, DimBlock>>>(d_input, d_output, dim[0], dim[1], dim[2]);
    } else if (perm[0] == 1 && perm[1] == 2 && perm[2] == 0) {
        transpose_shm_kernel_tmpl<1, 2, 0>
            <<<DimGrid, DimBlock>>>(d_input, d_output, dim[0], dim[1], dim[2]);
    } else if (perm[0] == 2 && perm[1] == 0 && perm[2] == 1) {
        transpose_shm_kernel_tmpl<2, 0, 1>
            <<<DimGrid, DimBlock>>>(d_input, d_output, dim[0], dim[1], dim[2]);
    } else if (perm[0] == 2 && perm[1] == 1 && perm[2] == 0) {
        transpose_shm_kernel_tmpl<2, 1, 0>
            <<<DimGrid, DimBlock>>>(d_input, d_output, dim[0], dim[1], dim[2]);
    }
}

// ============================================================================
// C++ SECTION
void array_init_rand(float* array, int size) {
    unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
    std::default_random_engine            generator(seed);
    std::uniform_real_distribution<float> distribution(0, 1);
    for (int i = 0; i < size; i++)
        array[i] = distribution(generator);
}

void array_init_seq(float* array, int size) {
    for (int i = 0; i < size; ++i)
        array[i] = i * 1.0f;
}

void array_print(const float* array, int size) {
    for (int i = 0; i < size; ++i) {
        std::cout << array[i] << ", ";
    }
    std::cout << std::endl;
}

void transpose_cpu(const float* input, float* output, const int* iDim,
                   const int* perm) {
    int oDim[] = {iDim[perm[0]], iDim[perm[1]], iDim[perm[2]]};
    for (int z = 0; z < iDim[0]; ++z) {
        for (int y = 0; y < iDim[1]; ++y) {
            for (int x = 0; x < iDim[2]; ++x) {
                int idx[]  = {z, y, x};
                int odx[]  = {idx[perm[0]], idx[perm[1]], idx[perm[2]]};
                int iIndex = (idx[0] * iDim[1] * iDim[2]) + (idx[1] * iDim[2]) +
                             (idx[2]);
                int oIndex = (odx[0] * oDim[1] * oDim[2]) + (odx[1] * oDim[2]) +
                             (odx[2]);
                output[oIndex] = input[iIndex];
            }
        }
    }
}

bool array_check(const float* gold, const float* result, int size) {
    for (int i = 0; i < size; ++i) {
        if (result[i] != gold[i]) {
            return false;
        }
    }
    return true;
}

void process(std::string name, bool testbench_mode, const float* gold,
             const float* result, int size, float kernel_ms, float host_ms,
             std::ofstream& file, bool print_speedup) {
    kernel_ms /= NUM_REPS;
    bool  is_correct = array_check(gold, result, size);
    float bandwidth  = 2 * size * sizeof(float) * 1e-6 / kernel_ms;
    float speedup    = host_ms / kernel_ms;

    if (ENABLE_PRINT) {
        array_print(gold, size);
        array_print(result, size);
    }

    if (!testbench_mode) {
        std::cout << name << std::endl;
        std::cout << "            Check: " << (is_correct ? "OK" : "FAIL")
                  << std::endl;
        if (print_speedup) {
            std::cout << "        Time (ms): " << kernel_ms << std::endl;
            std::cout << "     Speedup (ms): " << speedup << "x" << std::endl;
        }
        std::cout << " Bandwidth (GB/s): " << bandwidth << std::endl;
        std::cout << std::endl;
    } else {
        file << name << std::endl;
        file << is_correct << std::endl;
        file << bandwidth << std::endl;
        file << speedup << std::endl;
    }
}

void print_info(int tile, int size, const int* dim, const int* perm,
                const dim3& DimBlock, const dim3& DimGrid, float host_ms) {
    std::cout << "tile:    " << tile << std::endl;
    std::cout << "size:    " << size << std::endl;
    std::cout << "dimension:   (" << dim[0] << ", " << dim[1] << ", " << dim[2]
              << ")" << std::endl;
    std::cout << "permutation: (" << perm[0] << ", " << perm[1] << ", "
              << perm[2] << ")" << std::endl;
    std::cout << "DimBlock:    (" << DimBlock.x << ", " << DimBlock.y << ", "
              << DimBlock.z << ")" << std::endl;
    std::cout << "DimGrid:     (" << DimGrid.x << ", " << DimGrid.y << ", "
              << DimGrid.z << ")" << std::endl;
    std::cout << "Host Time (ms): " << host_ms << std::endl;
    std::cout << std::endl;
}


// ============================================================================
// MAIN
int main(int argc, char* argv[]) {
    // ------------------------------------------------------------------------
    // GET ARGS
    if (argc < 5) {
        std::cout << "call: executable permutation dim_z dim_y dim_x"
                  << std::endl;
        std::cout << "example: ./testbench.out 120 32 32 32" << std::endl;
        return 0;
    }
    int testbench_mode = false;
    if (argc == 6 && std::string(argv[5]) == "testbench") {
        testbench_mode = true;
    }

    // ------------------------------------------------------------------------
    // GET INFO
    int dim[3]  = {std::stoi(argv[2]), std::stoi(argv[3]), std::stoi(argv[4])};
    int perm[3] = {argv[1][0] - 48, argv[1][1] - 48, argv[1][2] - 48};
    int size    = dim[0] * dim[1] * dim[2];
    const int  bytes = size * sizeof(float);
    const dim3 DimBlock(TILE, TILE, TILE);
    const dim3 DimGrid(std::ceil((float)dim[2] / DimBlock.x),
                       std::ceil((float)dim[1] / DimBlock.y),
                       std::ceil((float)dim[0] / DimBlock.z));

    // ------------------------------------------------------------------------
    // SETUP TIMERS
    float       host_ms, device_ms, kernel_ms;
    hipEvent_t startEvent, stopEvent;
    CHECK_CUDA(hipEventCreate(&startEvent));
    CHECK_CUDA(hipEventCreate(&stopEvent));
    std::chrono::steady_clock::time_point start;
    std::chrono::steady_clock::time_point end;

    // ------------------------------------------------------------------------
    // HOST MEMORY ALLOCATION AND INITIALIZATION
    float* h_input  = new float[size]{};
    float* h_output = new float[size]{};
    float* h_gold   = new float[size]{};

    // array_init_rand(h_input, size);
    array_init_seq(h_input, size);

    // ------------------------------------------------------------------------
    // HOST EXECUTION
    start = std::chrono::steady_clock::now();
    for (int i = 0; i < NUM_REPS; ++i)
        transpose_cpu(h_input, h_gold, dim, perm);
    end     = std::chrono::steady_clock::now();
    host_ms = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start)
                  .count() *
              1e-6 / NUM_REPS;

    // ------------------------------------------------------------------------
    // PRINT INFO
    std::ofstream log;
    if (!testbench_mode) {
        print_info(TILE, size, dim, perm, DimBlock, DimGrid, host_ms);
    } else {
        std::string log_name = "logs_kernel/";
        log_name += std::to_string(dim[0]) + "x" + std::to_string(dim[1]) +
                    "x" + std::to_string(dim[2]);
        log_name += "_";
        log_name += std::to_string(perm[0]) + std::to_string(perm[1]) +
                    std::to_string(perm[2]);
        log_name += ".log";
        log.open(log_name, std::ios::out);
    }

    // ------------------------------------------------------------------------
    // DEVICE MEMORY ALLOCATION AND INITIALIZATION
    float *d_input, *d_output;
    CHECK_CUDA(hipMalloc(&d_input, bytes));
    CHECK_CUDA(hipMalloc(&d_output, bytes));

    CHECK_CUDA(hipMemcpy(d_input, h_input, bytes, hipMemcpyHostToDevice));

    // ------------------------------------------------------------------------
    // COPY BANDWIDTH SIMPLE
    CHECK_CUDA(hipMemset(d_output, 0, bytes));  // Initialize output
    copy_simple_kernel<<<DimGrid, DimBlock>>>(d_input, d_output, dim[0], dim[1],
                                              dim[2]);  // warmup
    CHECK_CUDA(hipEventRecord(startEvent, 0));
    for (int i = 0; i < NUM_REPS; ++i)
        copy_simple_kernel<<<DimGrid, DimBlock>>>(d_input, d_output, dim[0],
                                                  dim[1], dim[2]);
    CHECK_CUDA(hipEventRecord(stopEvent, 0));
    CHECK_CUDA(hipEventSynchronize(stopEvent));
    CHECK_CUDA(hipEventElapsedTime(&kernel_ms, startEvent, stopEvent));
    CHECK_CUDA(hipMemcpy(h_output, d_output, bytes, hipMemcpyDeviceToHost));
    process("Copy simple", testbench_mode, h_input, h_output, size, kernel_ms,
            host_ms, log, false);

    // ------------------------------------------------------------------------
    // COPY BANDWIDTH SHARED-MEMORY
    CHECK_CUDA(hipMemset(d_output, 0, bytes));  // Initialize output
    copy_shm_kernel<<<DimGrid, DimBlock>>>(d_input, d_output, dim[0], dim[1],
                                           dim[2]);  // warmup
    CHECK_CUDA(hipEventRecord(startEvent, 0));
    for (int i = 0; i < NUM_REPS; ++i)
        copy_shm_kernel<<<DimGrid, DimBlock>>>(d_input, d_output, dim[0],
                                               dim[1], dim[2]);
    CHECK_CUDA(hipEventRecord(stopEvent, 0));
    CHECK_CUDA(hipEventSynchronize(stopEvent));
    CHECK_CUDA(hipEventElapsedTime(&kernel_ms, startEvent, stopEvent));
    CHECK_CUDA(hipMemcpy(h_output, d_output, bytes, hipMemcpyDeviceToHost));
    process("Copy with shared-memory", testbench_mode, h_input, h_output, size,
            kernel_ms, host_ms, log, false);

    // ------------------------------------------------------------------------
    // TRANSPOSE SIMPLE
    // CHECK_CUDA(cudaMemset(d_output, 0, bytes));  // Initialize output
    // transpose_simple_kernel<<<DimGrid, DimBlock>>>(
    //     d_input, d_output, dim[0], dim[1], dim[2], perm[0], perm[1],
    //     perm[2]);  // warmup
    // CHECK_CUDA(cudaEventRecord(startEvent, 0));
    // for (int i = 0; i < NUM_REPS; ++i)
    //     transpose_simple_kernel<<<DimGrid, DimBlock>>>(
    //         d_input, d_output, dim[0], dim[1], dim[2], perm[0], perm[1],
    //         perm[2]);
    // CHECK_CUDA(cudaEventRecord(stopEvent, 0));
    // CHECK_CUDA(cudaEventSynchronize(stopEvent));
    // CHECK_CUDA(cudaEventElapsedTime(&kernel_ms, startEvent, stopEvent));
    // CHECK_CUDA(cudaMemcpy(h_output, d_output, bytes,
    // cudaMemcpyDeviceToHost)); process("Transpose simple", testbench_mode,
    // h_gold, h_output, size,
    //         kernel_ms, host_ms, log, true);

    // ------------------------------------------------------------------------
    // TRANSPOSE SIMPLE SELECTOR
    // CHECK_CUDA(cudaMemset(d_output, 0, bytes));  // Initialize output
    // transpose_simple_selector(DimGrid, DimBlock, d_input, d_output, dim,
    //                           perm);  // warmup
    // CHECK_CUDA(cudaEventRecord(startEvent, 0));
    // for (int i = 0; i < NUM_REPS; ++i)
    //     transpose_simple_selector(DimGrid, DimBlock, d_input, d_output, dim,
    //                               perm);
    // CHECK_CUDA(cudaEventRecord(stopEvent, 0));
    // CHECK_CUDA(cudaEventSynchronize(stopEvent));
    // CHECK_CUDA(cudaEventElapsedTime(&kernel_ms, startEvent, stopEvent));
    // CHECK_CUDA(cudaMemcpy(h_output, d_output, bytes,
    // cudaMemcpyDeviceToHost)); process("Transpose simple selector",
    // testbench_mode, h_gold, h_output, size,
    //         kernel_ms, host_ms, log, true);


    // ------------------------------------------------------------------------
    // TRANSPOSE SHARED-MEMORY
    CHECK_CUDA(hipMemset(d_output, 0, bytes));  // Initialize output
    transpose_shm_kernel<<<DimGrid, DimBlock>>>(
        d_input, d_output, dim[0], dim[1], dim[2], perm[0], perm[1],
        perm[2]);  // warmup
    CHECK_CUDA(hipEventRecord(startEvent, 0));
    for (int i = 0; i < NUM_REPS; ++i)
        transpose_shm_kernel<<<DimGrid, DimBlock>>>(d_input, d_output, dim[0],
                                                    dim[1], dim[2], perm[0],
                                                    perm[1], perm[2]);
    CHECK_CUDA(hipEventRecord(stopEvent, 0));
    CHECK_CUDA(hipEventSynchronize(stopEvent));
    CHECK_CUDA(hipEventElapsedTime(&kernel_ms, startEvent, stopEvent));
    CHECK_CUDA(hipMemcpy(h_output, d_output, bytes, hipMemcpyDeviceToHost));
    process("Transpose with shared-memory", testbench_mode, h_gold, h_output,
            size, kernel_ms, host_ms, log, true);

    // ------------------------------------------------------------------------
    // TRANSPOSE SHARED-MEMORY SELECTOR
    CHECK_CUDA(hipMemset(d_output, 0, bytes));  // Initialize output
    transpose_shm_selector(DimGrid, DimBlock, d_input, d_output, dim,
                           perm);  // warmup
    CHECK_CUDA(hipEventRecord(startEvent, 0));
    for (int i = 0; i < NUM_REPS; ++i)
        transpose_shm_selector(DimGrid, DimBlock, d_input, d_output, dim, perm);
    CHECK_CUDA(hipEventRecord(stopEvent, 0));
    CHECK_CUDA(hipEventSynchronize(stopEvent));
    CHECK_CUDA(hipEventElapsedTime(&kernel_ms, startEvent, stopEvent));
    CHECK_CUDA(hipMemcpy(h_output, d_output, bytes, hipMemcpyDeviceToHost));
    process("Transpose with shared-memory selector", testbench_mode, h_gold,
            h_output, size, kernel_ms, host_ms, log, true);

    // ------------------------------------------------------------------------
    // TRANSPOSE SHARED-MEMORY SELECTOR TMPL
    CHECK_CUDA(hipMemset(d_output, 0, bytes));  // Initialize output
    transpose_shm_tmpl_selector(DimGrid, DimBlock, d_input, d_output, dim,
                                perm);  // warmup
    CHECK_CUDA(hipEventRecord(startEvent, 0));
    for (int i = 0; i < NUM_REPS; ++i)
        transpose_shm_tmpl_selector(DimGrid, DimBlock, d_input, d_output, dim,
                                    perm);
    CHECK_CUDA(hipEventRecord(stopEvent, 0));
    CHECK_CUDA(hipEventSynchronize(stopEvent));
    CHECK_CUDA(hipEventElapsedTime(&kernel_ms, startEvent, stopEvent));
    CHECK_CUDA(hipMemcpy(h_output, d_output, bytes, hipMemcpyDeviceToHost));
    process("Transpose with shared-memory selector TMPL", testbench_mode,
            h_gold, h_output, size, kernel_ms, host_ms, log, true);

    // ------------------------------------------------------------------------
    // TRANSPOSE SHARED-MEMORY + BANK CONFLICT FREE
    // CHECK_CUDA(cudaMemset(d_output, 0, bytes));  // Initialize output
    // transpose_shm_bank_kernel<<<DimGrid, DimBlock>>>(
    //     d_input, d_output, dim[0], dim[1], dim[2], perm[0], perm[1],
    //     perm[2]);  // warmup
    // CHECK_CUDA(cudaEventRecord(startEvent, 0));
    // for (int i = 0; i < NUM_REPS; ++i)
    //     transpose_shm_bank_kernel<<<DimGrid, DimBlock>>>(
    //         d_input, d_output, dim[0], dim[1], dim[2], perm[0], perm[1],
    //         perm[2]);
    // CHECK_CUDA(cudaEventRecord(stopEvent, 0));
    // CHECK_CUDA(cudaEventSynchronize(stopEvent));
    // CHECK_CUDA(cudaEventElapsedTime(&kernel_ms, startEvent, stopEvent));
    // CHECK_CUDA(cudaMemcpy(h_output, d_output, bytes,
    // cudaMemcpyDeviceToHost)); process("Transpose with shared-memory (bank
    // conflict free)", testbench_mode,
    //         h_gold, h_output, size, kernel_ms, host_ms, log, true);

    // ------------------------------------------------------------------------
    // CLEAN SHUTDOWN
    log.close();
    delete[] h_input;
    delete[] h_output;
    delete[] h_gold;

    CHECK_CUDA(hipEventDestroy(startEvent));
    CHECK_CUDA(hipEventDestroy(stopEvent));
    CHECK_CUDA(hipFree(d_input));
    CHECK_CUDA(hipFree(d_output));
    hipDeviceReset();

    return 0;
}
