
#include <hip/hip_runtime.h>
#include <cassert>
#include <chrono>
#include <fstream>
#include <iostream>
#include <random>


// ============================================================================
// SETTINGS
const int  NUM_REPS     = 100;
const bool ENABLE_PRINT = false;

const int TILE3D    = 8;
const int TILE2D_v1 = 32;
const int TILE2D_v2 = 8;


// ============================================================================
// CUDA SECTION
inline hipError_t CHECK_CUDA(hipError_t result) {
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
}

// COPY
__global__ void copy3d_simple_kernel(const float* d_input, float* d_output,
                                     int dimz, int dimy, int dimx) {
    int x     = blockIdx.x * blockDim.x + threadIdx.x;
    int y     = blockIdx.y * blockDim.y + threadIdx.y;
    int z     = blockIdx.z * blockDim.z + threadIdx.z;
    int index = (z * dimy * dimx) + (y * dimx) + x;

    if (z < dimz && y < dimy && x < dimx) {
        d_output[index] = d_input[index];
    }
}

__global__ void copy2d_simple_kernel(const float* d_input, float* d_output,
                                     const int dimy, const int dimx) {
    int x     = blockIdx.x * blockDim.x + threadIdx.x;
    int y     = blockIdx.y * blockDim.y + threadIdx.y;
    int index = y * dimx + x;

    if (y < dimy && x < dimx) {
        d_output[index] = d_input[index];
    }
}

// TRANSPOSE SIMPLE
template<int pz, int py, int px>
__global__ void transpose3d_simple_kerneltmplt(const float* d_input,
                                               float* d_output, int dimz,
                                               int dimy, int dimx) {
    int idx[3]  = {blockIdx.z * blockDim.z + threadIdx.z,
                  blockIdx.y * blockDim.y + threadIdx.y,
                  blockIdx.x * blockDim.x + threadIdx.x};
    int iDim[3] = {dimz, dimy, dimx};
    int oDim[3] = {iDim[pz], iDim[py], iDim[px]};
    int odx[3]  = {idx[pz], idx[py], idx[px]};
    int iIndex  = (idx[0] * iDim[1] * iDim[2]) + (idx[1] * iDim[2]) + idx[2];
    int oIndex  = (odx[0] * oDim[1] * oDim[2]) + (odx[1] * oDim[2]) + odx[2];

    if (idx[0] < dimz && idx[1] < dimy && idx[2] < dimx) {
        d_output[oIndex] = d_input[iIndex];
    }
}

void transpose3d_simple_selector(const dim3& DimGrid, const dim3& DimBlock,
                                 const float* d_input, float* d_output,
                                 const int* dim, const int* perm) {
    if (perm[0] == 0 && perm[1] == 1 && perm[2] == 2) {
        transpose3d_simple_kerneltmplt<0, 1, 2>
            <<<DimGrid, DimBlock>>>(d_input, d_output, dim[0], dim[1], dim[2]);
    } else if (perm[0] == 0 && perm[1] == 2 && perm[2] == 1) {
        transpose3d_simple_kerneltmplt<0, 2, 1>
            <<<DimGrid, DimBlock>>>(d_input, d_output, dim[0], dim[1], dim[2]);
    } else if (perm[0] == 1 && perm[1] == 0 && perm[2] == 2) {
        transpose3d_simple_kerneltmplt<1, 0, 2>
            <<<DimGrid, DimBlock>>>(d_input, d_output, dim[0], dim[1], dim[2]);
    } else if (perm[0] == 1 && perm[1] == 2 && perm[2] == 0) {
        transpose3d_simple_kerneltmplt<1, 2, 0>
            <<<DimGrid, DimBlock>>>(d_input, d_output, dim[0], dim[1], dim[2]);
    } else if (perm[0] == 2 && perm[1] == 0 && perm[2] == 1) {
        transpose3d_simple_kerneltmplt<2, 0, 1>
            <<<DimGrid, DimBlock>>>(d_input, d_output, dim[0], dim[1], dim[2]);
    } else if (perm[0] == 2 && perm[1] == 1 && perm[2] == 0) {
        transpose3d_simple_kerneltmplt<2, 1, 0>
            <<<DimGrid, DimBlock>>>(d_input, d_output, dim[0], dim[1], dim[2]);
    }
}

__global__ void transpose2d_simple_kernel(const float* d_input, float* d_output,
                                          const int dimy, const int dimx) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (y < dimy && x < dimx) {
        d_output[x * dimy + y] = d_input[y * dimx + x];
    }
}

// TRANSPOSE SHARED MEMORY
template<int pz, int py, int px>
__global__ void transpose3d_shm_kerneltmplt(const float* d_input,
                                            float* d_output, int dimz, int dimy,
                                            int dimx) {
    __shared__ float buffer[TILE3D][TILE3D][TILE3D];

    int iDim[3] = {dimz, dimy, dimx};
    int x       = blockIdx.x * TILE3D + threadIdx.x;
    int y       = blockIdx.y * TILE3D + threadIdx.y;
    int z       = blockIdx.z * TILE3D + threadIdx.z;
    if (z < iDim[0] && y < iDim[1] && x < iDim[2]) {
        int iIndex     = (z * iDim[1] * iDim[2]) + (y * iDim[2]) + x;
        int threads[3] = {threadIdx.z, threadIdx.y, threadIdx.x};
        buffer[threads[pz]][threads[py]][threads[px]] = d_input[iIndex];
    }
    __syncthreads();

    int oDim[3]   = {iDim[pz], iDim[py], iDim[px]};
    int blocks[3] = {blockIdx.z, blockIdx.y, blockIdx.x};
    x             = blocks[px] * TILE3D + threadIdx.x;
    y             = blocks[py] * TILE3D + threadIdx.y;
    z             = blocks[pz] * TILE3D + threadIdx.z;
    if (z < oDim[0] && y < oDim[1] && x < oDim[2]) {
        int oIndex       = (z * oDim[1] * oDim[2]) + (y * oDim[2]) + x;
        d_output[oIndex] = buffer[threadIdx.z][threadIdx.y][threadIdx.x];
    }
}

void transpose3d_shm_selector(const dim3& DimGrid, const dim3& DimBlock,
                              const float* d_input, float* d_output,
                              const int* dim, const int* perm) {
    if (perm[0] == 0 && perm[1] == 1 && perm[2] == 2) {
        transpose3d_shm_kerneltmplt<0, 1, 2>
            <<<DimGrid, DimBlock>>>(d_input, d_output, dim[0], dim[1], dim[2]);
    } else if (perm[0] == 0 && perm[1] == 2 && perm[2] == 1) {
        transpose3d_shm_kerneltmplt<0, 2, 1>
            <<<DimGrid, DimBlock>>>(d_input, d_output, dim[0], dim[1], dim[2]);
    } else if (perm[0] == 1 && perm[1] == 0 && perm[2] == 2) {
        transpose3d_shm_kerneltmplt<1, 0, 2>
            <<<DimGrid, DimBlock>>>(d_input, d_output, dim[0], dim[1], dim[2]);
    } else if (perm[0] == 1 && perm[1] == 2 && perm[2] == 0) {
        transpose3d_shm_kerneltmplt<1, 2, 0>
            <<<DimGrid, DimBlock>>>(d_input, d_output, dim[0], dim[1], dim[2]);
    } else if (perm[0] == 2 && perm[1] == 0 && perm[2] == 1) {
        transpose3d_shm_kerneltmplt<2, 0, 1>
            <<<DimGrid, DimBlock>>>(d_input, d_output, dim[0], dim[1], dim[2]);
    } else if (perm[0] == 2 && perm[1] == 1 && perm[2] == 0) {
        transpose3d_shm_kerneltmplt<2, 1, 0>
            <<<DimGrid, DimBlock>>>(d_input, d_output, dim[0], dim[1], dim[2]);
    }
}

__global__ void transpose2d_shm_kernel_v1(const float* d_input, float* d_output,
                                          const int dimy, const int dimx) {
    __shared__ float buffer[TILE2D_v1][TILE2D_v1];

    int x = blockIdx.x * TILE2D_v1 + threadIdx.x;
    int y = blockIdx.y * TILE2D_v1 + threadIdx.y;
    if (y < dimy && x < dimx) {
        buffer[threadIdx.y][threadIdx.x] = d_input[y * dimx + x];
    }
    __syncthreads();

    x = blockIdx.y * TILE2D_v1 + threadIdx.x;
    y = blockIdx.x * TILE2D_v1 + threadIdx.y;
    if (y < dimx && x < dimy) {
        d_output[y * dimy + x] = buffer[threadIdx.x][threadIdx.y];
    }
}

__global__ void transpose2d_shm_kernel_v2(const float* d_input, float* d_output,
                                          const int dimy, const int dimx) {
    __shared__ float buffer[TILE2D_v2][TILE2D_v2];

    int x = blockIdx.x * TILE2D_v2 + threadIdx.x;
    int y = blockIdx.y * TILE2D_v2 + threadIdx.y;
    if (y < dimy && x < dimx) {
        buffer[threadIdx.y][threadIdx.x] = d_input[y * dimx + x];
    }
    __syncthreads();

    x = blockIdx.y * TILE2D_v2 + threadIdx.x;
    y = blockIdx.x * TILE2D_v2 + threadIdx.y;
    if (y < dimx && x < dimy) {
        d_output[y * dimy + x] = buffer[threadIdx.x][threadIdx.y];
    }
}

// TRANSPOSE SHARED MEMORY + BANK CONFLICT FREE
template<int pz, int py, int px, int tilez, int tiley, int tilex>
__global__ void transpose3d_shm_bank_kerneltmplt(const float* d_input,
                                                 float* d_output, int dimz,
                                                 int dimy, int dimx) {
    __shared__ float buffer[tilez][tiley][tilex];

    int iDim[3] = {dimz, dimy, dimx};
    int x       = blockIdx.x * TILE3D + threadIdx.x;
    int y       = blockIdx.y * TILE3D + threadIdx.y;
    int z       = blockIdx.z * TILE3D + threadIdx.z;
    if (z < iDim[0] && y < iDim[1] && x < iDim[2]) {
        int iIndex     = (z * iDim[1] * iDim[2]) + (y * iDim[2]) + x;
        int threads[3] = {threadIdx.z, threadIdx.y, threadIdx.x};
        buffer[threads[pz]][threads[py]][threads[px]] = d_input[iIndex];
    }
    __syncthreads();

    int oDim[3]   = {iDim[pz], iDim[py], iDim[px]};
    int blocks[3] = {blockIdx.z, blockIdx.y, blockIdx.x};
    x             = blocks[px] * TILE3D + threadIdx.x;
    y             = blocks[py] * TILE3D + threadIdx.y;
    z             = blocks[pz] * TILE3D + threadIdx.z;
    if (z < oDim[0] && y < oDim[1] && x < oDim[2]) {
        int oIndex       = (z * oDim[1] * oDim[2]) + (y * oDim[2]) + x;
        d_output[oIndex] = buffer[threadIdx.z][threadIdx.y][threadIdx.x];
    }
}

void transpose3d_shm_bank_selector(const dim3& DimGrid, const dim3& DimBlock,
                                   const float* d_input, float* d_output,
                                   const int* dim, const int* perm) {
    if (perm[0] == 0 && perm[1] == 1 && perm[2] == 2) {
        transpose3d_shm_bank_kerneltmplt<0, 1, 2, 8, 8, 8>
            <<<DimGrid, DimBlock>>>(d_input, d_output, dim[0], dim[1], dim[2]);
    } else if (perm[0] == 0 && perm[1] == 2 && perm[2] == 1) {
        transpose3d_shm_bank_kerneltmplt<0, 2, 1, 8, 8, 12>
            <<<DimGrid, DimBlock>>>(d_input, d_output, dim[0], dim[1], dim[2]);
    } else if (perm[0] == 1 && perm[1] == 0 && perm[2] == 2) {
        transpose3d_shm_bank_kerneltmplt<1, 0, 2, 8, 9, 8>
            <<<DimGrid, DimBlock>>>(d_input, d_output, dim[0], dim[1], dim[2]);
    } else if (perm[0] == 1 && perm[1] == 2 && perm[2] == 0) {
        transpose3d_shm_bank_kerneltmplt<1, 2, 0, 8, 10, 10>
            <<<DimGrid, DimBlock>>>(d_input, d_output, dim[0], dim[1], dim[2]);
    } else if (perm[0] == 2 && perm[1] == 0 && perm[2] == 1) {
        transpose3d_shm_bank_kerneltmplt<2, 0, 1, 8, 8, 9>
            <<<DimGrid, DimBlock>>>(d_input, d_output, dim[0], dim[1], dim[2]);
    } else if (perm[0] == 2 && perm[1] == 1 && perm[2] == 0) {
        transpose3d_shm_bank_kerneltmplt<2, 1, 0, 8, 12, 9>
            <<<DimGrid, DimBlock>>>(d_input, d_output, dim[0], dim[1], dim[2]);
    }
}

__global__ void transpose2d_shm_bank_kernel_v1(const float* d_input,
                                               float* d_output, const int dimy,
                                               const int dimx) {
    __shared__ float buffer[TILE2D_v1][TILE2D_v1 + 1];

    int x = blockIdx.x * TILE2D_v1 + threadIdx.x;
    int y = blockIdx.y * TILE2D_v1 + threadIdx.y;
    if (y < dimy && x < dimx) {
        buffer[threadIdx.y][threadIdx.x] = d_input[y * dimx + x];
    }
    __syncthreads();

    x = blockIdx.y * TILE2D_v1 + threadIdx.x;
    y = blockIdx.x * TILE2D_v1 + threadIdx.y;
    if (y < dimx && x < dimy) {
        d_output[y * dimy + x] = buffer[threadIdx.x][threadIdx.y];
    }
}
__global__ void transpose2d_shm_bank_kernel_v2(const float* d_input,
                                               float* d_output, const int dimy,
                                               const int dimx) {
    __shared__ float buffer[TILE2D_v2][TILE2D_v2 + 1];

    int x = blockIdx.x * TILE2D_v2 + threadIdx.x;
    int y = blockIdx.y * TILE2D_v2 + threadIdx.y;
    if (y < dimy && x < dimx) {
        buffer[threadIdx.y][threadIdx.x] = d_input[y * dimx + x];
    }
    __syncthreads();

    x = blockIdx.y * TILE2D_v2 + threadIdx.x;
    y = blockIdx.x * TILE2D_v2 + threadIdx.y;
    if (y < dimx && x < dimy) {
        d_output[y * dimy + x] = buffer[threadIdx.x][threadIdx.y];
    }
}


// ============================================================================
// C++ SECTION
void array_init_rand(float* array, int size) {
    unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
    std::default_random_engine            generator(seed);
    std::uniform_real_distribution<float> distribution(0, 1);
    for (int i = 0; i < size; i++)
        array[i] = distribution(generator);
}

void array_init_seq(float* array, int size) {
    for (int i = 0; i < size; ++i)
        array[i] = i * 1.0f;
}

void array_print(const float* array, int size) {
    for (int i = 0; i < size; ++i) {
        std::cout << array[i] << ", ";
    }
    std::cout << std::endl;
}

void transpose_cpu(const float* matrix, float* result, int m, int n) {
    for (int row = 0; row < m; ++row) {
        for (int col = 0; col < n; ++col) {
            result[col * m + row] = matrix[row * n + col];
        }
    }
}

bool array_check(const float* gold, const float* result, int size) {
    for (int i = 0; i < size; ++i) {
        if (result[i] != gold[i]) {
            return false;
        }
    }
    return true;
}

void process(std::string name, bool testbench_mode, const float* gold,
             const float* result, int size, float kernel_ms, float host_ms,
             std::ofstream& file, bool print_speedup) {
    kernel_ms /= NUM_REPS;
    bool  is_correct = array_check(gold, result, size);
    float bandwidth  = 2 * size * sizeof(float) * 1e-6 / kernel_ms;
    float speedup    = host_ms / kernel_ms;

    if (ENABLE_PRINT) {
        array_print(gold, size);
        array_print(result, size);
    }

    if (!testbench_mode) {
        std::cout << name << std::endl;
        std::cout << "            Check: " << (is_correct ? "OK" : "FAIL")
                  << std::endl;
        if (print_speedup) {
            std::cout << "        Time (ms): " << kernel_ms << std::endl;
            std::cout << "     Speedup (ms): " << speedup << "x" << std::endl;
        }
        std::cout << " Bandwidth (GB/s): " << bandwidth << std::endl;
        std::cout << std::endl;
    } else {
        file << name << std::endl;
        file << is_correct << std::endl;
        file << kernel_ms << std::endl;
        file << speedup << std::endl;
        file << bandwidth << std::endl;
    }
}


// ============================================================================
// MAIN
int main(int argc, char* argv[]) {
    // ------------------------------------------------------------------------
    // GET ARGS
    if (argc < 3) {
        std::cout << "call: executable dim_y dim_x" << std::endl;
        std::cout << "example: ./demo_matrix.out 32 32" << std::endl;
        return 0;
    }
    int testbench_mode = false;
    if (argc == 4 && std::string(argv[3]) == "testbench") {
        testbench_mode = true;
    }

    // ------------------------------------------------------------------------
    // GET INFO
    int        dim_y   = std::stoi(argv[1]);
    int        dim_x   = std::stoi(argv[2]);
    int        dim[3]  = {1, dim_y, dim_x};
    int        perm[3] = {0, 2, 1};
    int        size    = dim_y * dim_x;
    const int  bytes   = size * sizeof(float);
    const dim3 DimBlock3D(TILE3D, TILE3D, TILE3D);
    const dim3 DimGrid3D(std::ceil((float)dim_x / DimBlock3D.x),
                         std::ceil((float)dim_y / DimBlock3D.y), 1);
    const dim3 DimBlock2D_v1(TILE2D_v1, TILE2D_v1, 1);
    const dim3 DimGrid2D_v1(std::ceil((float)dim_x / DimBlock2D_v1.x),
                            std::ceil((float)dim_y / DimBlock2D_v1.y), 1);
    const dim3 DimBlock2D_v2(TILE2D_v2, TILE2D_v2, 1);
    const dim3 DimGrid2D_v2(std::ceil((float)dim_x / DimBlock2D_v2.x),
                            std::ceil((float)dim_y / DimBlock2D_v2.y), 1);

    std::string str_tile2d_v1 =
        "(" + std::to_string(TILE2D_v1) + "," + std::to_string(TILE2D_v1) + ")";
    std::string str_tile2d_v2 =
        "(" + std::to_string(TILE2D_v2) + "," + std::to_string(TILE2D_v2) + ")";
    std::string str_tile3d = "(" + std::to_string(TILE3D) + "," +
                             std::to_string(TILE3D) + "," +
                             std::to_string(TILE3D) + ")";

    // ------------------------------------------------------------------------
    // SETUP TIMERS
    float       host_ms, device_ms, kernel_ms;
    hipEvent_t startEvent, stopEvent;
    CHECK_CUDA(hipEventCreate(&startEvent));
    CHECK_CUDA(hipEventCreate(&stopEvent));
    std::chrono::steady_clock::time_point start;
    std::chrono::steady_clock::time_point end;

    // ------------------------------------------------------------------------
    // HOST MEMORY ALLOCATION AND INITIALIZATION
    float* h_input  = new float[size]{};
    float* h_output = new float[size]{};
    float* h_gold   = new float[size]{};

    // array_init_rand(h_input, size);
    array_init_seq(h_input, size);

    // ------------------------------------------------------------------------
    // HOST EXECUTION
    start = std::chrono::steady_clock::now();
    for (int i = 0; i < NUM_REPS; ++i)
        transpose_cpu(h_input, h_gold, dim_y, dim_x);
    end     = std::chrono::steady_clock::now();
    host_ms = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start)
                  .count() *
              1e-6 / NUM_REPS;

    // ------------------------------------------------------------------------
    // PRINT INFO
    std::ofstream log;
    if (!testbench_mode) {
        std::cout << "size:    " << size << std::endl;
        std::cout << "dimension:   (" << dim_y << ", " << dim_x << ")"
                  << std::endl;
        std::cout << "DimBlock 2D V1:    (" << DimBlock2D_v1.x << ", "
                  << DimBlock2D_v1.y << ", " << DimBlock2D_v1.z << ")"
                  << std::endl;
        std::cout << "DimGrid 2D V1:     (" << DimGrid2D_v1.x << ", "
                  << DimGrid2D_v1.y << ", " << DimGrid2D_v1.z << ")"
                  << std::endl;
        std::cout << "DimBlock 2D V2:    (" << DimBlock2D_v2.x << ", "
                  << DimBlock2D_v2.y << ", " << DimBlock2D_v2.z << ")"
                  << std::endl;
        std::cout << "DimGrid 2D V2:     (" << DimGrid2D_v2.x << ", "
                  << DimGrid2D_v2.y << ", " << DimGrid2D_v2.z << ")"
                  << std::endl;
        std::cout << "DimBlock 3D:    (" << DimBlock3D.x << ", " << DimBlock3D.y
                  << ", " << DimBlock3D.z << ")" << std::endl;
        std::cout << "DimGrid 3D:     (" << DimGrid3D.x << ", " << DimGrid3D.y
                  << ", " << DimGrid3D.z << ")" << std::endl;
        std::cout << "Host Time (ms): " << host_ms << std::endl;
        std::cout << std::endl;
    } else {
        std::string log_name = "logs/logs_matrix/";
        log_name += std::to_string(dim_y) + "x" + std::to_string(dim_x);
        log_name += ".log";
        log.open(log_name, std::ios::out);
        log << host_ms << std::endl;
    }

    // ------------------------------------------------------------------------
    // DEVICE MEMORY ALLOCATION AND INITIALIZATION
    float *d_input, *d_output;
    CHECK_CUDA(hipMalloc(&d_input, bytes));
    CHECK_CUDA(hipMalloc(&d_output, bytes));

    CHECK_CUDA(hipMemcpy(d_input, h_input, bytes, hipMemcpyHostToDevice));

    // ========================================================================
    // COPY 2D BANDWIDTH SIMPLE V1
    CHECK_CUDA(hipMemset(d_output, 0, bytes));  // Initialize output
    copy2d_simple_kernel<<<DimGrid2D_v1, DimBlock2D_v1>>>(d_input, d_output,
                                                          dim_y,
                                                          dim_x);  // warmup
    CHECK_CUDA(hipEventRecord(startEvent, 0));
    for (int i = 0; i < NUM_REPS; ++i)
        copy2d_simple_kernel<<<DimGrid2D_v1, DimBlock2D_v1>>>(d_input, d_output,
                                                              dim_y, dim_x);
    CHECK_CUDA(hipEventRecord(stopEvent, 0));
    CHECK_CUDA(hipEventSynchronize(stopEvent));
    CHECK_CUDA(hipEventElapsedTime(&kernel_ms, startEvent, stopEvent));
    CHECK_CUDA(hipMemcpy(h_output, d_output, bytes, hipMemcpyDeviceToHost));
    process("Copy 2D simple " + str_tile2d_v1, testbench_mode, h_input,
            h_output, size, kernel_ms, host_ms, log, false);

    // ------------------------------------------------------------------------
    // TRANSPOSE 2D SIMPLE V1
    CHECK_CUDA(hipMemset(d_output, 0, bytes));  // Initialize output
    transpose2d_simple_kernel<<<DimGrid2D_v1, DimBlock2D_v1>>>(
        d_input, d_output, dim_y,
        dim_x);  // warmup
    CHECK_CUDA(hipEventRecord(startEvent, 0));
    for (int i = 0; i < NUM_REPS; ++i)
        transpose2d_simple_kernel<<<DimGrid2D_v1, DimBlock2D_v1>>>(
            d_input, d_output, dim_y, dim_x);
    CHECK_CUDA(hipEventRecord(stopEvent, 0));
    CHECK_CUDA(hipEventSynchronize(stopEvent));
    CHECK_CUDA(hipEventElapsedTime(&kernel_ms, startEvent, stopEvent));
    CHECK_CUDA(hipMemcpy(h_output, d_output, bytes, hipMemcpyDeviceToHost));
    process("Transpose 2D simple " + str_tile2d_v1, testbench_mode, h_gold,
            h_output, size, kernel_ms, host_ms, log, true);

    // ------------------------------------------------------------------------
    // TRANSPOSE 2D SHARED-MEMORY V1
    CHECK_CUDA(hipMemset(d_output, 0, bytes));  // Initialize output
    transpose2d_shm_kernel_v1<<<DimGrid2D_v1, DimBlock2D_v1>>>(
        d_input, d_output, dim_y,
        dim_x);  // warmup
    CHECK_CUDA(hipEventRecord(startEvent, 0));
    for (int i = 0; i < NUM_REPS; ++i)
        transpose2d_shm_kernel_v1<<<DimGrid2D_v1, DimBlock2D_v1>>>(
            d_input, d_output, dim_y, dim_x);
    CHECK_CUDA(hipEventRecord(stopEvent, 0));
    CHECK_CUDA(hipEventSynchronize(stopEvent));
    CHECK_CUDA(hipEventElapsedTime(&kernel_ms, startEvent, stopEvent));
    CHECK_CUDA(hipMemcpy(h_output, d_output, bytes, hipMemcpyDeviceToHost));
    process("Transpose 2D with shared-memory " + str_tile2d_v1, testbench_mode,
            h_gold, h_output, size, kernel_ms, host_ms, log, true);

    // ------------------------------------------------------------------------
    // TRANSPOSE 2D SHARED-MEMORY + BANK CONFLICT FREE V1
    CHECK_CUDA(hipMemset(d_output, 0, bytes));  // Initialize output
    transpose2d_shm_bank_kernel_v1<<<DimGrid2D_v1, DimBlock2D_v1>>>(
        d_input, d_output, dim_y,
        dim_x);  // warmup
    CHECK_CUDA(hipEventRecord(startEvent, 0));
    for (int i = 0; i < NUM_REPS; ++i)
        transpose2d_shm_bank_kernel_v1<<<DimGrid2D_v1, DimBlock2D_v1>>>(
            d_input, d_output, dim_y, dim_x);
    CHECK_CUDA(hipEventRecord(stopEvent, 0));
    CHECK_CUDA(hipEventSynchronize(stopEvent));
    CHECK_CUDA(hipEventElapsedTime(&kernel_ms, startEvent, stopEvent));
    CHECK_CUDA(hipMemcpy(h_output, d_output, bytes, hipMemcpyDeviceToHost));
    process(
        "Transpose 2D with shared-memory (bank conflict free) " + str_tile2d_v1,
        testbench_mode, h_gold, h_output, size, kernel_ms, host_ms, log, true);

    // ========================================================================
    // COPY 2D BANDWIDTH SIMPLE V2
    CHECK_CUDA(hipMemset(d_output, 0, bytes));  // Initialize output
    copy2d_simple_kernel<<<DimGrid2D_v2, DimBlock2D_v2>>>(d_input, d_output,
                                                          dim_y,
                                                          dim_x);  // warmup
    CHECK_CUDA(hipEventRecord(startEvent, 0));
    for (int i = 0; i < NUM_REPS; ++i)
        copy2d_simple_kernel<<<DimGrid2D_v2, DimBlock2D_v2>>>(d_input, d_output,
                                                              dim_y, dim_x);
    CHECK_CUDA(hipEventRecord(stopEvent, 0));
    CHECK_CUDA(hipEventSynchronize(stopEvent));
    CHECK_CUDA(hipEventElapsedTime(&kernel_ms, startEvent, stopEvent));
    CHECK_CUDA(hipMemcpy(h_output, d_output, bytes, hipMemcpyDeviceToHost));
    process("Copy 2D simple " + str_tile2d_v2, testbench_mode, h_input,
            h_output, size, kernel_ms, host_ms, log, false);

    // ------------------------------------------------------------------------
    // TRANSPOSE 2D SIMPLE V2
    CHECK_CUDA(hipMemset(d_output, 0, bytes));  // Initialize output
    transpose2d_simple_kernel<<<DimGrid2D_v2, DimBlock2D_v2>>>(
        d_input, d_output, dim_y,
        dim_x);  // warmup
    CHECK_CUDA(hipEventRecord(startEvent, 0));
    for (int i = 0; i < NUM_REPS; ++i)
        transpose2d_simple_kernel<<<DimGrid2D_v2, DimBlock2D_v2>>>(
            d_input, d_output, dim_y, dim_x);
    CHECK_CUDA(hipEventRecord(stopEvent, 0));
    CHECK_CUDA(hipEventSynchronize(stopEvent));
    CHECK_CUDA(hipEventElapsedTime(&kernel_ms, startEvent, stopEvent));
    CHECK_CUDA(hipMemcpy(h_output, d_output, bytes, hipMemcpyDeviceToHost));
    process("Transpose 2D simple " + str_tile2d_v2, testbench_mode, h_gold,
            h_output, size, kernel_ms, host_ms, log, true);

    // ------------------------------------------------------------------------
    // TRANSPOSE 2D SHARED-MEMORY V2
    CHECK_CUDA(hipMemset(d_output, 0, bytes));  // Initialize output
    transpose2d_shm_kernel_v2<<<DimGrid2D_v2, DimBlock2D_v2>>>(
        d_input, d_output, dim_y,
        dim_x);  // warmup
    CHECK_CUDA(hipEventRecord(startEvent, 0));
    for (int i = 0; i < NUM_REPS; ++i)
        transpose2d_shm_kernel_v2<<<DimGrid2D_v2, DimBlock2D_v2>>>(
            d_input, d_output, dim_y, dim_x);
    CHECK_CUDA(hipEventRecord(stopEvent, 0));
    CHECK_CUDA(hipEventSynchronize(stopEvent));
    CHECK_CUDA(hipEventElapsedTime(&kernel_ms, startEvent, stopEvent));
    CHECK_CUDA(hipMemcpy(h_output, d_output, bytes, hipMemcpyDeviceToHost));
    process("Transpose 2D with shared-memory " + str_tile2d_v2, testbench_mode,
            h_gold, h_output, size, kernel_ms, host_ms, log, true);

    // ------------------------------------------------------------------------
    // TRANSPOSE 2D SHARED-MEMORY + BANK CONFLICT FREE V2
    CHECK_CUDA(hipMemset(d_output, 0, bytes));  // Initialize output
    transpose2d_shm_bank_kernel_v2<<<DimGrid2D_v2, DimBlock2D_v2>>>(
        d_input, d_output, dim_y,
        dim_x);  // warmup
    CHECK_CUDA(hipEventRecord(startEvent, 0));
    for (int i = 0; i < NUM_REPS; ++i)
        transpose2d_shm_bank_kernel_v2<<<DimGrid2D_v2, DimBlock2D_v2>>>(
            d_input, d_output, dim_y, dim_x);
    CHECK_CUDA(hipEventRecord(stopEvent, 0));
    CHECK_CUDA(hipEventSynchronize(stopEvent));
    CHECK_CUDA(hipEventElapsedTime(&kernel_ms, startEvent, stopEvent));
    CHECK_CUDA(hipMemcpy(h_output, d_output, bytes, hipMemcpyDeviceToHost));
    process(
        "Transpose 2D with shared-memory (bank conflict free) " + str_tile2d_v2,
        testbench_mode, h_gold, h_output, size, kernel_ms, host_ms, log, true);

    // ========================================================================
    // COPY 3D BANDWIDTH SIMPLE
    CHECK_CUDA(hipMemset(d_output, 0, bytes));  // Initialize output
    copy3d_simple_kernel<<<DimGrid3D, DimBlock3D>>>(d_input, d_output, 1, dim_y,
                                                    dim_x);  // warmup
    CHECK_CUDA(hipEventRecord(startEvent, 0));
    for (int i = 0; i < NUM_REPS; ++i)
        copy3d_simple_kernel<<<DimGrid3D, DimBlock3D>>>(d_input, d_output, 1,
                                                        dim_y, dim_x);
    CHECK_CUDA(hipEventRecord(stopEvent, 0));
    CHECK_CUDA(hipEventSynchronize(stopEvent));
    CHECK_CUDA(hipEventElapsedTime(&kernel_ms, startEvent, stopEvent));
    CHECK_CUDA(hipMemcpy(h_output, d_output, bytes, hipMemcpyDeviceToHost));
    process("Copy 3D simple " + str_tile3d, testbench_mode, h_input, h_output,
            size, kernel_ms, host_ms, log, false);

    // ------------------------------------------------------------------------
    // TRANSPOSE 3D SIMPLE
    CHECK_CUDA(hipMemset(d_output, 0, bytes));  // Initialize output
    transpose3d_simple_selector(DimGrid3D, DimBlock3D, d_input, d_output, dim,
                                perm);  // warmup
    CHECK_CUDA(hipEventRecord(startEvent, 0));
    for (int i = 0; i < NUM_REPS; ++i)
        transpose3d_simple_selector(DimGrid3D, DimBlock3D, d_input, d_output,
                                    dim, perm);
    CHECK_CUDA(hipEventRecord(stopEvent, 0));
    CHECK_CUDA(hipEventSynchronize(stopEvent));
    CHECK_CUDA(hipEventElapsedTime(&kernel_ms, startEvent, stopEvent));
    CHECK_CUDA(hipMemcpy(h_output, d_output, bytes, hipMemcpyDeviceToHost));
    process("Transpose 3D simple " + str_tile3d, testbench_mode, h_gold,
            h_output, size, kernel_ms, host_ms, log, true);

    // ------------------------------------------------------------------------
    // TRANSPOSE 3D SHARED-MEMORY
    CHECK_CUDA(hipMemset(d_output, 0, bytes));  // Initialize output
    transpose3d_shm_selector(DimGrid3D, DimBlock3D, d_input, d_output, dim,
                             perm);  // warmup
    CHECK_CUDA(hipEventRecord(startEvent, 0));
    for (int i = 0; i < NUM_REPS; ++i)
        transpose3d_shm_selector(DimGrid3D, DimBlock3D, d_input, d_output, dim,
                                 perm);
    CHECK_CUDA(hipEventRecord(stopEvent, 0));
    CHECK_CUDA(hipEventSynchronize(stopEvent));
    CHECK_CUDA(hipEventElapsedTime(&kernel_ms, startEvent, stopEvent));
    CHECK_CUDA(hipMemcpy(h_output, d_output, bytes, hipMemcpyDeviceToHost));
    process("Transpose shared-memory template " + str_tile3d, testbench_mode,
            h_gold, h_output, size, kernel_ms, host_ms, log, true);

    // ------------------------------------------------------------------------
    // TRANSPOSE 3D SHARED-MEMORY + BANK CONFLICT FREE
    CHECK_CUDA(hipMemset(d_output, 0, bytes));  // Initialize output
    transpose3d_shm_bank_selector(DimGrid3D, DimBlock3D, d_input, d_output, dim,
                                  perm);  // warmup
    CHECK_CUDA(hipEventRecord(startEvent, 0));
    for (int i = 0; i < NUM_REPS; ++i)
        transpose3d_shm_bank_selector(DimGrid3D, DimBlock3D, d_input, d_output,
                                      dim, perm);
    CHECK_CUDA(hipEventRecord(stopEvent, 0));
    CHECK_CUDA(hipEventSynchronize(stopEvent));
    CHECK_CUDA(hipEventElapsedTime(&kernel_ms, startEvent, stopEvent));
    CHECK_CUDA(hipMemcpy(h_output, d_output, bytes, hipMemcpyDeviceToHost));
    process("Transpose shared-memory bank-conflict-free template " + str_tile3d,
            testbench_mode, h_gold, h_output, size, kernel_ms, host_ms, log,
            true);


    // ------------------------------------------------------------------------
    // CLEAN SHUTDOWN
    log.close();
    delete[] h_input;
    delete[] h_output;
    delete[] h_gold;

    CHECK_CUDA(hipEventDestroy(startEvent));
    CHECK_CUDA(hipEventDestroy(stopEvent));
    CHECK_CUDA(hipFree(d_input));
    CHECK_CUDA(hipFree(d_output));
    hipDeviceReset();

    return 0;
}
