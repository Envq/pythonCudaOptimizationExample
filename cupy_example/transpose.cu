
#include <hip/hip_runtime.h>
#include <stdio.h>


const int TILE = 32;


extern "C" __global__ void transpose(const float* d_input, float* d_output,
                                     const int m, const int n) {
    __shared__ float buffer[TILE][TILE + 1];

    // read matrix in linear order
    int col = blockIdx.x * TILE + threadIdx.x;
    int row = blockIdx.y * TILE + threadIdx.y;
    if ((col < n) && (row < m)) {
        buffer[threadIdx.y][threadIdx.x] = d_input[row * n + col];
    }
    __syncthreads();

    // write transposed matrix in linear order
    col = blockIdx.y * TILE + threadIdx.x;
    row = blockIdx.x * TILE + threadIdx.y;
    if ((col < m) && (row < n)) {
        // transpose is done with buffer
        d_output[row * m + col] = buffer[threadIdx.x][threadIdx.y];
    }
}
